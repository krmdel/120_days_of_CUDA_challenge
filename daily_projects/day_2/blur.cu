#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLUR_SIZE 1

__global__ void blur_kernel(unsigned char* image, unsigned char* blurred, unsigned int width, unsigned int height) {
    int outrow = blockIdx.y * blockDim.y + threadIdx.y;
    int outcol = blockIdx.x * blockDim.x + threadIdx.x;

    if (outrow < height && outcol < width) {
        unsigned int average = 0;
        for (int inrow = outrow - BLUR_SIZE; inrow < outrow + BLUR_SIZE + 1; inrow++) {
            for (int incol = outcol - BLUR_SIZE; incol < outcol + BLUR_SIZE + 1; incol++) {
                if (inrow >= 0 && inrow < height && incol >= 0 && incol < width) {
                    average += image[inrow * width + incol];
                }
            }
        }
        blurred[outrow * width + outcol] = average / ((2 * BLUR_SIZE + 1) * (2 * BLUR_SIZE + 1));
    }
}

void blur_gpu(unsigned char* image, unsigned char* blurred, unsigned int width, unsigned int height) {

    // Create CUDA events for timing different stages
    hipEvent_t start_alloc, stop_alloc;
    hipEvent_t start_h2d, stop_h2d;
    hipEvent_t start_kernel, stop_kernel;
    hipEvent_t start_d2h, stop_d2h;
    float time_alloc, time_h2d, time_kernel, time_d2h;

    hipEventCreate(&start_alloc);
    hipEventCreate(&stop_alloc);
    hipEventCreate(&start_h2d);
    hipEventCreate(&stop_h2d);
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventCreate(&start_d2h);
    hipEventCreate(&stop_d2h);

    int size = width * height;
    unsigned char *image_d, *blurred_d;

    // Allocating memory on the device
    hipEventRecord(start_alloc, 0);
    hipMalloc((void**)&image_d, size * sizeof(unsigned char));
    hipMalloc((void**)&blurred_d, size * sizeof(unsigned char));
    hipDeviceSynchronize();
    hipEventRecord(stop_alloc, 0);
    hipEventSynchronize(stop_alloc);
    hipEventElapsedTime(&time_alloc, start_alloc, stop_alloc);
    printf("Time for memory allocation: %f ms\n", time_alloc);

    // Copying data from CPU to GPU
    hipEventRecord(start_h2d, 0);
    hipMemcpy(image_d, image, size * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    hipEventRecord(stop_h2d, 0);
    hipEventSynchronize(stop_h2d);
    hipEventElapsedTime(&time_h2d, start_h2d, stop_h2d);
    printf("Time for host-to-device copy: %f ms\n", time_h2d);

    // Calling the kernel and performing the operation
    hipEventRecord(start_kernel, 0);
    dim3 numThreadsPerBlock(32, 32);
    dim3 numBlocks((width + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x,
                   (height + numThreadsPerBlock.y - 1) / numThreadsPerBlock.y);
    blur_kernel<<<numBlocks, numThreadsPerBlock>>>(image_d, blurred_d, width, height);
    hipDeviceSynchronize();
    hipEventRecord(stop_kernel, 0);
    hipEventSynchronize(stop_kernel);
    hipEventElapsedTime(&time_kernel, start_kernel, stop_kernel);
    printf("Time for kernel execution: %f ms\n", time_kernel);

    // Copying data from GPU to CPU
    hipEventRecord(start_d2h, 0);
    hipMemcpy(blurred, blurred_d, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop_d2h, 0);
    hipEventSynchronize(stop_d2h);
    hipEventElapsedTime(&time_d2h, start_d2h, stop_d2h);
    printf("Time for device-to-host copy: %f ms\n", time_d2h);

    // Free GPU memory
    hipFree(image_d);
    hipFree(blurred_d);
    hipDeviceSynchronize();

    // Clean up events
    hipEventDestroy(start_alloc);
    hipEventDestroy(stop_alloc);
    hipEventDestroy(start_h2d);
    hipEventDestroy(stop_h2d);
    hipEventDestroy(start_kernel);
    hipEventDestroy(stop_kernel);
    hipEventDestroy(start_d2h);
    hipEventDestroy(stop_d2h);
}

int main() {

    unsigned int width = 1024;
    unsigned int height = 1024;
    int size = width * height;

    // Allocate memory on CPU
    unsigned char* image = (unsigned char*)malloc(size * sizeof(unsigned char));
    unsigned char* blurred = (unsigned char*)malloc(size * sizeof(unsigned char));

    // Initialize host arrays
    for (int i = 0; i < size; i++) {
        image[i] = 255;  // Example: white image.
    }

    // Call the function to blur image
    blur_gpu(image, blurred, width, height);

    // Free CPU memory
    free(image);
    free(blurred);

    return 0;
}
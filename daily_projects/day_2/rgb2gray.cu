#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void rgb2gray_kernel(unsigned char* red, unsigned char* green, unsigned char* blue, unsigned char* gray, unsigned int width, unsigned int height) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width) {
        unsigned int i = row * width + col;
        gray[i] = 3/10 * red[i] + 6/10 * green[i] + 1/10 * blue[i];
    }
}

void rgb2gray_gpu(unsigned char* red, unsigned char* green, unsigned char* blue, unsigned char* gray, unsigned int width, unsigned int height) {
    
    // Create CUDA events for timing different stages
    hipEvent_t start_alloc, stop_alloc;
    hipEvent_t start_h2d, stop_h2d;
    hipEvent_t start_kernel, stop_kernel;
    hipEvent_t start_d2h, stop_d2h;
    float time_alloc, time_h2d, time_kernel, time_d2h;

    hipEventCreate(&start_alloc);
    hipEventCreate(&stop_alloc);
    hipEventCreate(&start_h2d);
    hipEventCreate(&stop_h2d);
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventCreate(&start_d2h);
    hipEventCreate(&stop_d2h);

    int size = width * height;
    unsigned char *red_d, *green_d, *blue_d, *gray_d;

    // Allocating memory on the device
    hipEventRecord(start_alloc, 0);
    hipMalloc((void**)&red_d, size * sizeof(unsigned char));
    hipMalloc((void**)&green_d, size * sizeof(unsigned char));
    hipMalloc((void**)&blue_d, size * sizeof(unsigned char));
    hipMalloc((void**)&gray_d, size * sizeof(unsigned char));
    hipDeviceSynchronize();
    hipEventRecord(stop_alloc, 0);
    hipEventSynchronize(stop_alloc);
    hipEventElapsedTime(&time_alloc, start_alloc, stop_alloc);
    printf("Time for memory allocation: %f ms\n", time_alloc);

    // Copying data from CPU to GPU
    hipEventRecord(start_h2d, 0);
    hipMemcpy(red_d, red, size * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(green_d, green, size * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(blue_d, blue, size * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    hipEventRecord(stop_h2d, 0);
    hipEventSynchronize(stop_h2d);
    hipEventElapsedTime(&time_h2d, start_h2d, stop_h2d);
    printf("Time for host-to-device copy: %f ms\n", time_h2d);

    // Calling the kernel and performing the operation
    hipEventRecord(start_kernel, 0);
    dim3 numThreadsPerBlock(32, 32);
    dim3 numBlocks((width + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x,
                   (height + numThreadsPerBlock.y - 1) / numThreadsPerBlock.y);
    rgb2gray_kernel<<<numBlocks, numThreadsPerBlock>>>(red_d, green_d, blue_d, gray_d, width, height);
    hipDeviceSynchronize();
    hipEventRecord(stop_kernel, 0);
    hipEventSynchronize(stop_kernel);
    hipEventElapsedTime(&time_kernel, start_kernel, stop_kernel);
    printf("Time for kernel execution: %f ms\n", time_kernel);

    // Copying data from GPU to CPU
    hipEventRecord(start_d2h, 0);
    hipMemcpy(gray, gray_d, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop_d2h, 0);
    hipEventSynchronize(stop_d2h);
    hipEventElapsedTime(&time_d2h, start_d2h, stop_d2h);
    printf("Time for device-to-host copy: %f ms\n", time_d2h);

    // Free GPU memory
    hipFree(red_d);
    hipFree(green_d);
    hipFree(blue_d);
    hipFree(gray_d);
    hipDeviceSynchronize();

    // Clean up events
    hipEventDestroy(start_alloc);
    hipEventDestroy(stop_alloc);
    hipEventDestroy(start_h2d);
    hipEventDestroy(stop_h2d);
    hipEventDestroy(start_kernel);
    hipEventDestroy(stop_kernel);
    hipEventDestroy(start_d2h);
    hipEventDestroy(stop_d2h);
}

int main() {

    unsigned int width = 1024;
    unsigned int height = 1024;
    int size = width * height;

    // Allocate memory on CPU
    unsigned char* red   = (unsigned char*)malloc(size * sizeof(unsigned char));
    unsigned char* green = (unsigned char*)malloc(size * sizeof(unsigned char));
    unsigned char* blue  = (unsigned char*)malloc(size * sizeof(unsigned char));
    unsigned char* gray  = (unsigned char*)malloc(size * sizeof(unsigned char));

    // Initialize host arrays
    for (int i = 0; i < size; i++) {
        red[i]   = 255;  // maximum intensity for red
        green[i] = 255;  // medium intensity for green
        blue[i] = 255;   // low intensity for blue
    }

    // Call the function to convert RGB to grayscale
    rgb2gray_gpu(red, green, blue, gray, width, height);

    // Free CPU memory
    free(red);
    free(green);
    free(blue);
    free(gray);

    return 0;
}
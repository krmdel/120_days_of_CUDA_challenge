/****************************************************************************************
 *  fir_naive.cu  – Day-82: FIR filtering (one thread per output sample)
 *
 *  Build :  nvcc -O3 -std=c++17 fir_naive.cu -o fir
 *  Run   :  ./fir  [N] [T]
 *           N … signal length     (default 1 048 576)
 *           T … number of taps    (default 63)
 *****************************************************************************************/
#include <hip/hip_runtime.h>

#include <vector>
#include <random>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <cstdlib>
#include <algorithm>        // std::max

// Error helpers
#define CUDA_CHECK(x)  do{ hipError_t rc=(x); if(rc!=hipSuccess){            \
    std::cerr<<"CUDA "<<hipGetErrorString(rc)<<" @"<<__FILE__<<":"<<__LINE__;\
    std::exit(EXIT_FAILURE);} }while(0)

// FIR kernel
__global__ void fir_naive_kernel(const float* __restrict__ x,
                                 const float* __restrict__ h,
                                 float*       __restrict__ y,
                                 int N, int T)
{
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if(n >= N) return;

    float acc = 0.f;
    #pragma unroll 4
    for(int k = 0; k < T; ++k){
        int idx = n - k;
        if(idx >= 0) acc += h[k] * x[idx];          // zero-padding for idx<0
    }
    y[n] = acc;
}

int main(int argc, char** argv)
{
    const int N = (argc > 1) ? std::atoi(argv[1]) : 1'048'576;   // 2^20
    const int T = (argc > 2) ? std::atoi(argv[2]) : 63;
    std::cout << "Naïve FIR  —  N = " << N << ",  taps = " << T << "\n\n";

    // Host data
    std::vector<float> h_sig(N), h_taps(T);
    std::mt19937 rng(0);
    std::uniform_real_distribution<float> dist(-1.f, 1.f);
    for(float& v : h_sig ) v = dist(rng);
    for(float& v : h_taps) v = dist(rng);

    // Device buffers
    float *d_sig, *d_taps, *d_out;
    CUDA_CHECK(hipMalloc(&d_sig , sizeof(float)*N));
    CUDA_CHECK(hipMalloc(&d_taps, sizeof(float)*T));
    CUDA_CHECK(hipMalloc(&d_out , sizeof(float)*N));

    // Timing events
    hipEvent_t eH2D0, eH2D1, eK0, eK1, eD2H0, eD2H1;
    for(auto& ev : {&eH2D0,&eH2D1,&eK0,&eK1,&eD2H0,&eD2H1})
        hipEventCreate(ev);

    // Host → Device
    hipEventRecord(eH2D0);
    CUDA_CHECK(hipMemcpy(d_sig , h_sig .data(), sizeof(float)*N, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_taps, h_taps.data(), sizeof(float)*T, hipMemcpyHostToDevice));
    hipEventRecord(eH2D1);

    // Kernel launch
    const int THREADS = 256;
    const int BLOCKS  = (N + THREADS - 1) / THREADS;
    hipEventRecord(eK0);
    fir_naive_kernel<<<BLOCKS, THREADS>>>(d_sig, d_taps, d_out, N, T);
    hipEventRecord(eK1);

    // Device → Host
    std::vector<float> h_out(N);
    hipEventRecord(eD2H0);
    CUDA_CHECK(hipMemcpy(h_out.data(), d_out, sizeof(float)*N, hipMemcpyDeviceToHost));
    hipEventRecord(eD2H1);
    CUDA_CHECK(hipDeviceSynchronize());

    // Timings
    float h2d_ms, ker_ms, d2h_ms;
    hipEventElapsedTime(&h2d_ms, eH2D0, eH2D1);
    hipEventElapsedTime(&ker_ms, eK0 , eK1 );
    hipEventElapsedTime(&d2h_ms, eD2H0, eD2H1);

    std::cout << std::fixed << std::setprecision(3);
    std::cout << "GPU H2D copy : " << h2d_ms << " ms\n";
    std::cout << "GPU Kernel   : " << ker_ms << " ms\n";
    std::cout << "GPU D2H copy : " << d2h_ms << " ms\n";
    std::cout << "GPU Total    : " << (h2d_ms + ker_ms + d2h_ms) << " ms\n";

    // Cleanup
    hipFree(d_sig); hipFree(d_taps); hipFree(d_out);
    for(auto ev : {eH2D0,eH2D1,eK0,eK1,eD2H0,eD2H1}) hipEventDestroy(ev);
    return 0;
}
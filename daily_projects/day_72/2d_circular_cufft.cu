#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <chrono>

#define CHECK_CUDA(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error: " << hipGetErrorString(err) << " at line " << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    }

#define CHECK_CUFFT(call) \
    { \
        hipfftResult err = call; \
        if (err != HIPFFT_SUCCESS) { \
            std::cerr << "CUFFT error: " << err << " at line " << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    }

// Kernel for element-wise complex multiplication
__global__ void complexPointwiseMul2D(hipfftComplex* a, const hipfftComplex* b, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height) {
        a[idx] = hipCmulf(a[idx], b[idx]);
    }
}

int main() {
    const int WIDTH = 128;
    const int HEIGHT = 128;
    const int N = WIDTH * HEIGHT;
    size_t size = sizeof(hipfftComplex) * N;
        std::cout << "Image " << WIDTH << "×" << HEIGHT
              << " (" << N << " px)\n\n";
    // Host inputs
    std::vector<float> x(N), h(N);
    for (int i = 0; i < HEIGHT; ++i) {
        for (int j = 0; j < WIDTH; ++j) {
            x[i * WIDTH + j] = sinf(2 * M_PI * i / HEIGHT) * cosf(2 * M_PI * j / WIDTH);
            h[i * WIDTH + j] = cosf(2 * M_PI * i / HEIGHT) * sinf(2 * M_PI * j / WIDTH);
        }
    }

    // Convert to complex
    std::vector<hipfftComplex> x_c(N), h_c(N);
    for (int i = 0; i < N; ++i) {
        x_c[i] = make_hipFloatComplex(x[i], 0.0f);
        h_c[i] = make_hipFloatComplex(h[i], 0.0f);
    }

    // Device memory
    hipfftComplex *d_x, *d_h;
    CHECK_CUDA(hipMalloc(&d_x, size));
    CHECK_CUDA(hipMalloc(&d_h, size));
    CHECK_CUDA(hipMemcpy(d_x, x_c.data(), size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_h, h_c.data(), size, hipMemcpyHostToDevice));

    // cuFFT plan
    hipfftHandle plan;
    CHECK_CUFFT(hipfftPlan2d(&plan, HEIGHT, WIDTH, HIPFFT_C2C));

    // Timing
    hipEvent_t start, stop;
    float elapsed;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Forward FFTs
    CHECK_CUFFT(hipfftExecC2C(plan, d_x, d_x, HIPFFT_FORWARD));
    CHECK_CUFFT(hipfftExecC2C(plan, d_h, d_h, HIPFFT_FORWARD));

    // Element-wise multiply
    dim3 threads(16, 16);
    dim3 blocks((WIDTH + threads.x - 1) / threads.x, (HEIGHT + threads.y - 1) / threads.y);
    complexPointwiseMul2D<<<blocks, threads>>>(d_x, d_h, WIDTH, HEIGHT);
    CHECK_CUDA(hipGetLastError());

    // Inverse FFT
    CHECK_CUFFT(hipfftExecC2C(plan, d_x, d_x, HIPFFT_BACKWARD));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);

    // Copy back and normalize
    std::vector<hipfftComplex> result(N);
    CHECK_CUDA(hipMemcpy(result.data(), d_x, size, hipMemcpyDeviceToHost));
    for (int i = 0; i < N; ++i) {
        result[i].x /= N;
        result[i].y /= N;
    }

    // Printing inference time
    std::cout << "\nInference time: " << elapsed << " ms\n";

    // Cleanup
    hipfftDestroy(plan);
    hipFree(d_x);
    hipFree(d_h);

    return 0;
}
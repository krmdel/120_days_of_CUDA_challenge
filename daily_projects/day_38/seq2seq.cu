#include "hip/hip_runtime.h"
// seq2seq_integration.cu
// Integrated CUDA and CPU implementation for a simple encoder-decoder (seq2seq) model.
// This code combines tokenizer (with embedding lookup), positional encoding,
// a single-layer encoder and a single-layer decoder.
// The parameters have been set to match your original encoder/decoder implementations:
//    - Sequence length (M/MAX_TOKENS): 512 tokens
//    - Model dimension (d_total/EMBEDDING_DIM): 256
//    - Number of attention heads (H): 4  (with d_head = 256/4 = 64)
//    - Feed-forward inner dimension (d_ff): 4*256 = 1024
//
// Both CPU and GPU inference paths are implemented and timed so you can compare them.

#include <iostream>
#include <sstream>
#include <vector>
#include <string>
#include <unordered_map>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>

// ---------------------------------------------------------------------
// Error checking macro
#define CHECK_CUDA(call) {                                 \
    hipError_t err = call;                                \
    if(err != hipSuccess) {                               \
        std::cerr << "CUDA error: " << hipGetErrorString(err) \
                  << " at " << __FILE__ << ":" << __LINE__  \
                  << std::endl;                            \
        exit(1);                                           \
    }                                                      \
}

// ---------------------------------------------------------------------
// Global Constants
// For the tokenizer and embedding lookup:
const int VOCAB_SIZE    = 4;       // same as before
const int NUM_SENTENCES = 1;       // assume one input sentence for encoder and one for decoder each
const int MAX_TOKENS    = 512;     // sequence length
const int EMBEDDING_DIM = 256;     // same as d_model

// For Encoder/Decoder (from your original encoder.cu/decoder.cu):
const int M = MAX_TOKENS;         // sequence length = 512
const int d_total = 256;          // model (hidden) dimension
const int H = 4;                  // number of attention heads
const int d_head = d_total / H;   // 64
const int d_ff = 4 * d_total;     // feed-forward inner dimension = 1024

// ---------------------------------------------------------------------
// GPU Kernels
// 1. Embedding Lookup Kernel (for tokenization)
__global__ void embedding_lookup_kernel(const int* token_ids, 
                                          const float* embedding_matrix, 
                                          float* output, 
                                          int total_tokens, 
                                          int embedding_dim) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx >= total_tokens) return;
    int token_id = token_ids[idx];
    for (int d = 0; d < embedding_dim; d++){
        if(token_id < 0 || token_id >= VOCAB_SIZE)
            output[idx * embedding_dim + d] = 0.0f;
        else
            output[idx * embedding_dim + d] = embedding_matrix[token_id * embedding_dim + d];
    }
}

// 2. Positional Encoding Kernel
__global__ void positional_encoding_kernel(float* pe, int seq_len, int d_model) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int total = seq_len * d_model;
    if(idx >= total) return;
    
    int pos = idx / d_model;
    int j   = idx % d_model;
    float factor = (j % 2 == 0) ? (j / 2.0f) : ((j - 1) / 2.0f);
    float div_term = expf(-logf(10000.0f) * factor / d_model);
    float angle = pos * div_term;
    pe[idx] = (j % 2 == 0) ? sinf(angle) : cosf(angle);
}

// 3. Tiled Matrix Multiplication Kernel (used for linear projections)
#define TILE_WIDTH 16
__global__ void matmul_kernel(const float *A, const float *B, float *C,
                              int M_mat, int K, int N) {
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    float sum = 0.0f;
    for (int t = 0; t < (K + TILE_WIDTH - 1) / TILE_WIDTH; t++) {
        if(row < M_mat && t * TILE_WIDTH + threadIdx.x < K)
            ds_A[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_WIDTH + threadIdx.x];
        else
            ds_A[threadIdx.y][threadIdx.x] = 0.0f;
            
        if(col < N && t * TILE_WIDTH + threadIdx.y < K)
            ds_B[threadIdx.y][threadIdx.x] = B[(t * TILE_WIDTH + threadIdx.y) * N + col];
        else
            ds_B[threadIdx.y][threadIdx.x] = 0.0f;
            
        __syncthreads();
        for (int i = 0; i < TILE_WIDTH; i++){
            sum += ds_A[threadIdx.y][i] * ds_B[i][threadIdx.x];
        }
        __syncthreads();
    }
    if(row < M_mat && col < N)
        C[row * N + col] = sum;
}

// 4. Compute Attention Scores Kernel (for multi-head attention)
// This kernel computes the scaled dot-product between Q and K.
__global__ void compute_scores_kernel(const float *Q, const float *K, float *d_scores,
                                        int M_val, int N_val, int H_val, int d_head) {
    int h = blockIdx.z; // head index
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y; // query index
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x; // key index

    __shared__ float sQ[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sK[TILE_WIDTH][TILE_WIDTH];

    float val = 0.0f;
    for (int t = 0; t < (d_head + TILE_WIDTH - 1) / TILE_WIDTH; t++) {
        int indexQ = t * TILE_WIDTH + threadIdx.x;
        int indexK = t * TILE_WIDTH + threadIdx.y;
        if (row < M_val && indexQ < d_head)
            sQ[threadIdx.y][threadIdx.x] = Q[row * (H_val*d_head) + h * d_head + indexQ];
        else
            sQ[threadIdx.y][threadIdx.x] = 0.0f;
        if (col < N_val && indexK < d_head)
            sK[threadIdx.y][threadIdx.x] = K[col * (H_val*d_head) + h * d_head + indexK];
        else
            sK[threadIdx.y][threadIdx.x] = 0.0f;
        __syncthreads();
        for (int i = 0; i < TILE_WIDTH; i++){
            val += sQ[threadIdx.y][i] * sK[i][threadIdx.x];
        }
        __syncthreads();
    }
    val = val / sqrtf((float)d_head);
    if (row < M_val && col < N_val)
        d_scores[((row * N_val) + col) * H_val + h] = val;
}

// 5. Softmax Kernel (operating per row of scores)
__global__ void softmax_kernel(float *d_scores, int M_val, int N_val, int H_val) {
    int idx = blockIdx.x; // one block per (query, head) pair; total blocks = M_val * H_val
    int i = idx / H_val;
    int h = idx % H_val;
    int t = threadIdx.x;
    extern __shared__ float shmem[];
    float thread_max = -1e20f;
    for (int j = t; j < N_val; j += blockDim.x) {
        float s = d_scores[((i * N_val) + j) * H_val + h];
        if(s > thread_max)
            thread_max = s;
    }
    shmem[t] = thread_max;
    __syncthreads();
    // Reduction for maximum
    for (int stride = blockDim.x/2; stride > 0; stride /= 2) {
        if(t < stride && (t + stride) < blockDim.x)
            shmem[t] = fmaxf(shmem[t], shmem[t + stride]);
        __syncthreads();
    }
    float max_val = shmem[0];
    __syncthreads();
    float sum_exp = 0.0f;
    for (int j = t; j < N_val; j += blockDim.x) {
        float exp_val = expf(d_scores[((i * N_val) + j) * H_val + h] - max_val);
        d_scores[((i * N_val) + j) * H_val + h] = exp_val;
        sum_exp += exp_val;
    }
    shmem[t] = sum_exp;
    __syncthreads();
    // Reduction for sum
    for (int stride = blockDim.x/2; stride > 0; stride /= 2) {
        if(t < stride)
            shmem[t] += shmem[t + stride];
        __syncthreads();
    }
    float total_exp = shmem[0];
    for (int j = t; j < N_val; j += blockDim.x) {
        d_scores[((i * N_val) + j) * H_val + h] /= total_exp;
    }
}

// 6. Weighted Sum Kernel: Computes attention output = scores * V.
__global__ void weighted_sum_kernel(const float *d_scores, const float *V, float *O,
                                      int M_val, int N_val, int H_val, int d_head) {
    int h = blockIdx.z;
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int k = blockIdx.x * TILE_WIDTH + threadIdx.x;
    if (row < M_val && k < d_head) {
        float sum_val = 0.0f;
        for (int j = 0; j < N_val; j++){
            float score = d_scores[((row * N_val) + j) * H_val + h];
            float v_val = V[j * (H_val * d_head) + h * d_head + k];
            sum_val += score * v_val;
        }
        O[row * (H_val * d_head) + h * d_head + k] = sum_val;
    }
}

// 7. Element-wise Addition Kernel: C = A + B.
__global__ void add_kernel(const float *A, const float *B, float *C, int total_elements) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < total_elements)
        C[idx] = A[idx] + B[idx];
}

// 8. ReLU Activation Kernel.
__global__ void relu_kernel(float *A, int total_elements) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < total_elements && A[idx] < 0)
        A[idx] = 0;
}

// 9. Layer Normalization Kernel (per row normalization).
__global__ void layer_norm_kernel(const float* input, float* output, int M_val, int N_val, float epsilon) {
    int row = blockIdx.x; // one block per row
    if(row < M_val) {
        extern __shared__ float shmem[];
        int tid = threadIdx.x;
        float sum = 0.0f;
        for (int j = tid; j < N_val; j += blockDim.x)
            sum += input[row * N_val + j];
        shmem[tid] = sum;
        __syncthreads();
        for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
            if(tid < stride)
                shmem[tid] += shmem[tid + stride];
            __syncthreads();
        }
        float mean = shmem[0] / N_val;
        __syncthreads();
        float var_sum = 0.0f;
        for (int j = tid; j < N_val; j += blockDim.x) {
            float diff = input[row * N_val + j] - mean;
            var_sum += diff * diff;
        }
        shmem[tid] = var_sum;
        __syncthreads();
        for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
            if(tid < stride)
                shmem[tid] += shmem[tid + stride];
            __syncthreads();
        }
        float var = shmem[0] / N_val;
        float inv_std = rsqrtf(var + epsilon);
        for (int j = tid; j < N_val; j += blockDim.x) {
            int idx = row * N_val + j;
            output[idx] = (input[idx] - mean) * inv_std;
        }
    }
}

// ---------------------------------------------------------------------
// CPU Utility Functions
// CPU Tokenizer: splits sentences by whitespace; pads to MAX_TOKENS.
void cpu_tokenizer(const std::vector<std::string>& sentences,
                   const std::unordered_map<std::string,int>& vocab,
                   std::vector<int>& token_ids) {
    int num_sentences = sentences.size();
    token_ids.assign(num_sentences * MAX_TOKENS, -1); // pad with -1
    for (int i = 0; i < num_sentences; i++) {
        std::istringstream iss(sentences[i]);
        std::string token;
        int token_idx = 0;
        while (iss >> token && token_idx < MAX_TOKENS) {
            if (vocab.find(token) != vocab.end())
                token_ids[i * MAX_TOKENS + token_idx] = vocab.at(token);
            else
                token_ids[i * MAX_TOKENS + token_idx] = -1;
            token_idx++;
        }
    }
}

// CPU Embedding Lookup: similar to the GPU kernel.
void cpu_embedding_lookup(const std::vector<int>& token_ids,
                          const std::vector<float>& embedding_matrix,
                          std::vector<float>& output, int total_tokens, int embedding_dim) {
    output.resize(total_tokens * embedding_dim);
    for (int i = 0; i < total_tokens; i++){
        int id = token_ids[i];
        for (int d = 0; d < embedding_dim; d++){
            if(id < 0 || id >= VOCAB_SIZE)
                output[i * embedding_dim + d] = 0.0f;
            else
                output[i * embedding_dim + d] = embedding_matrix[id * embedding_dim + d];
        }
    }
}

// CPU Positional Encoding
void cpu_positional_encoding(std::vector<float>& pe, int seq_len, int d_model) {
    pe.resize(seq_len * d_model);
    for (int pos = 0; pos < seq_len; pos++){
        for (int j = 0; j < d_model; j++){
            float factor = (j % 2 == 0) ? (j / 2.0f) : ((j - 1) / 2.0f);
            float div_term = exp(-log(10000.0f) * factor / d_model);
            float angle = pos * div_term;
            pe[pos * d_model + j] = (j % 2 == 0) ? sin(angle) : cos(angle);
        }
    }
}

// Naïve CPU Matrix Multiplication (A [M x K] * B [K x N] = C [M x N])
void cpu_matmul(const std::vector<float>& A, const std::vector<float>& B,
                std::vector<float>& C, int M_mat, int K, int N) {
    C.resize(M_mat * N, 0.0f);
    for (int i = 0; i < M_mat; i++){
        for (int j = 0; j < N; j++){
            float sum = 0.0f;
            for (int k = 0; k < K; k++){
                sum += A[i*K + k] * B[k*N + j];
            }
            C[i*N + j] = sum;
        }
    }
}

// CPU Multi-Head Attention (single-head version extended over H heads)
void cpu_multi_head_attention(const float *Q, const float *K, const float *V,
                              float *O, int M_val, int N_val, int H_val, int d_head) {
    // For each head h, for each query i, compute attention over N keys
    for (int h = 0; h < H_val; h++){
        for (int i = 0; i < M_val; i++){
            std::vector<float> scores(N_val, 0.0f);
            float max_val = -1e20f;
            for (int j = 0; j < N_val; j++){
                float dot = 0.0f;
                for (int k = 0; k < d_head; k++){
                    float q_val = Q[i * (H_val*d_head) + h*d_head + k];
                    float k_val = K[j * (H_val*d_head) + h*d_head + k];
                    dot += q_val * k_val;
                }
                dot /= sqrt((float)d_head);
                scores[j] = dot;
                if (dot > max_val)
                    max_val = dot;
            }
            float sum_exp = 0.0f;
            for (int j = 0; j < N_val; j++){
                scores[j] = exp(scores[j] - max_val);
                sum_exp += scores[j];
            }
            for (int j = 0; j < N_val; j++){
                scores[j] /= sum_exp;
            }
            for (int k = 0; k < d_head; k++){
                float val = 0.0f;
                for (int j = 0; j < N_val; j++){
                    float v_val = V[j * (H_val*d_head) + h*d_head + k];
                    val += scores[j] * v_val;
                }
                O[i * (H_val*d_head) + h*d_head + k] = val;
            }
        }
    }
}

// CPU Element-wise Addition
void cpu_add(const float *A, const float *B, float *C, int total_elements) {
    for (int i = 0; i < total_elements; i++){
        C[i] = A[i] + B[i];
    }
}

// CPU ReLU Activation
void cpu_relu(float *A, int total_elements) {
    for (int i = 0; i < total_elements; i++){
        if (A[i] < 0)
            A[i] = 0;
    }
}

// CPU Layer Normalization (per row normalization)
void cpu_layer_norm(const float* input, float* output, int M_val, int N_val, float epsilon) {
    for (int i = 0; i < M_val; i++){
        float sum = 0.0f;
        for (int j = 0; j < N_val; j++){
            sum += input[i*N_val + j];
        }
        float mean = sum / N_val;
        float var = 0.0f;
        for (int j = 0; j < N_val; j++){
            float diff = input[i*N_val + j] - mean;
            var += diff * diff;
        }
        var /= N_val;
        float inv_std = 1.0f / sqrt(var + epsilon);
        for (int j = 0; j < N_val; j++){
            output[i*N_val + j] = (input[i*N_val + j] - mean) * inv_std;
        }
    }
}

// ---------------------------------------------------------------------
// CPU Encoder Implementation
// Applies linear projections, multi-head self-attention and a feed-forward network.
void cpu_encoder(const std::vector<float>& X, std::vector<float>& output,
                 const std::vector<float>& Wq, const std::vector<float>& Wk,
                 const std::vector<float>& Wv, const std::vector<float>& Wo,
                 const std::vector<float>& W1, const std::vector<float>& W2) {
    std::vector<float> Q, K, V;
    cpu_matmul(X, Wq, Q, M, d_total, d_total);
    cpu_matmul(X, Wk, K, M, d_total, d_total);
    cpu_matmul(X, Wv, V, M, d_total, d_total);

    std::vector<float> attn(M * d_total, 0.0f);
    cpu_multi_head_attention(Q.data(), K.data(), V.data(), attn.data(), M, M, H, d_head);

    std::vector<float> MHA(M * d_total, 0.0f);
    cpu_matmul(attn, Wo, MHA, M, d_total, d_total);

    std::vector<float> add1(M * d_total, 0.0f);
    cpu_add(X.data(), MHA.data(), add1.data(), M * d_total);

    std::vector<float> ln1(M * d_total, 0.0f);
    cpu_layer_norm(add1.data(), ln1.data(), M, d_total, 1e-5);

    std::vector<float> ffn1(M * d_ff, 0.0f);
    cpu_matmul(ln1, W1, ffn1, M, d_total, d_ff);
    cpu_relu(ffn1.data(), M * d_ff);
    std::vector<float> ffn2(M * d_total, 0.0f);
    cpu_matmul(ffn1, W2, ffn2, M, d_ff, d_total);

    std::vector<float> add2(M * d_total, 0.0f);
    cpu_add(ln1.data(), ffn2.data(), add2.data(), M * d_total);

    std::vector<float> ln2(M * d_total, 0.0f);
    cpu_layer_norm(add2.data(), ln2.data(), M, d_total, 1e-5);

    output = ln2;
}

// ---------------------------------------------------------------------
// CPU Decoder Implementation
// A simplified one-layer decoder with self-attention, encoder-decoder attention, and feed-forward network.
void cpu_decoder(const std::vector<float>& Y, const std::vector<float>& enc_out,
                 std::vector<float>& output,
                 const std::vector<float>& Wq_self, const std::vector<float>& Wk_self,
                 const std::vector<float>& Wv_self, const std::vector<float>& Wo_self,
                 const std::vector<float>& Wq_encdec, const std::vector<float>& Wk_encdec,
                 const std::vector<float>& Wv_encdec, const std::vector<float>& Wo_encdec,
                 const std::vector<float>& W1, const std::vector<float>& W2) {
    // Self-Attention Block
    std::vector<float> Q_self, K_self, V_self;
    cpu_matmul(Y, Wq_self, Q_self, M, d_total, d_total);
    cpu_matmul(Y, Wk_self, K_self, M, d_total, d_total);
    cpu_matmul(Y, Wv_self, V_self, M, d_total, d_total);

    std::vector<float> attn_self(M * d_total, 0.0f);
    cpu_multi_head_attention(Q_self.data(), K_self.data(), V_self.data(), attn_self.data(), M, M, H, d_head);

    std::vector<float> MHA_self(M * d_total, 0.0f);
    cpu_matmul(attn_self, Wo_self, MHA_self, M, d_total, d_total);
    std::vector<float> add_self(M * d_total, 0.0f);
    cpu_add(Y.data(), MHA_self.data(), add_self.data(), M * d_total);
    std::vector<float> ln_self(M * d_total, 0.0f);
    cpu_layer_norm(add_self.data(), ln_self.data(), M, d_total, 1e-5);

    // Encoder-Decoder Attention Block
    std::vector<float> Q_encdec, K_encdec, V_encdec;
    cpu_matmul(ln_self, Wq_encdec, Q_encdec, M, d_total, d_total);
    cpu_matmul(enc_out,    Wk_encdec, K_encdec, M, d_total, d_total);
    cpu_matmul(enc_out,    Wv_encdec, V_encdec, M, d_total, d_total);
    std::vector<float> attn_encdec(M * d_total, 0.0f);
    cpu_multi_head_attention(Q_encdec.data(), K_encdec.data(), V_encdec.data(), attn_encdec.data(), M, M, H, d_head);
    std::vector<float> MHA_encdec(M * d_total, 0.0f);
    cpu_matmul(attn_encdec, Wo_encdec, MHA_encdec, M, d_total, d_total);
    std::vector<float> add_encdec(M * d_total, 0.0f);
    cpu_add(ln_self.data(), MHA_encdec.data(), add_encdec.data(), M * d_total);
    std::vector<float> ln_encdec(M * d_total, 0.0f);
    cpu_layer_norm(add_encdec.data(), ln_encdec.data(), M, d_total, 1e-5);

    // Feed-Forward Network Block
    std::vector<float> ffn1(M * d_ff, 0.0f);
    cpu_matmul(ln_encdec, W1, ffn1, M, d_total, d_ff);
    cpu_relu(ffn1.data(), M * d_ff);
    std::vector<float> ffn2(M * d_total, 0.0f);
    cpu_matmul(ffn1, W2, ffn2, M, d_ff, d_total);
    std::vector<float> add_ffn(M * d_total, 0.0f);
    cpu_add(ln_encdec.data(), ffn2.data(), add_ffn.data(), M * d_total);
    std::vector<float> ln_dec(M * d_total, 0.0f);
    cpu_layer_norm(add_ffn.data(), ln_dec.data(), M, d_total, 1e-5);

    output = ln_dec;
}

// ---------------------------------------------------------------------
// Main Function: Full integration of tokenizer, positional encoding, encoder and decoder.
// The program runs both CPU and GPU inference for comparison.
int main() {
    // -------------------------
    // 1. Tokenization
    // For demonstration, we use one short sentence for encoder and one for decoder.
    std::vector<std::string> encoder_sentences = {"hello world"};
    std::vector<std::string> decoder_sentences = {"cuda programming"};
    std::unordered_map<std::string,int> vocab = { {"hello", 0}, {"world", 1}, {"cuda", 2}, {"programming", 3} };

    std::vector<int> encoder_token_ids, decoder_token_ids;
    cpu_tokenizer(encoder_sentences, vocab, encoder_token_ids);
    cpu_tokenizer(decoder_sentences, vocab, decoder_token_ids);
    int total_tokens_enc = NUM_SENTENCES * MAX_TOKENS;
    int total_tokens_dec = NUM_SENTENCES * MAX_TOKENS;

    // -------------------------
    // 2. Embedding Lookup
    // Create an embedding matrix for the vocabulary of size VOCAB_SIZE x EMBEDDING_DIM.
    // For reproducibility we fill with a fixed pattern.
    std::vector<float> embedding_matrix(VOCAB_SIZE * EMBEDDING_DIM);
    for (int i = 0; i < VOCAB_SIZE; i++){
        for (int j = 0; j < EMBEDDING_DIM; j++){
            embedding_matrix[i * EMBEDDING_DIM + j] = static_cast<float>((i*17 + j + 1) % 100) / 100.0f;
        }
    }

    // CPU embedding lookup for encoder and decoder tokens.
    std::vector<float> enc_embeddings, dec_embeddings;
    cpu_embedding_lookup(encoder_token_ids, embedding_matrix, enc_embeddings, total_tokens_enc, EMBEDDING_DIM);
    cpu_embedding_lookup(decoder_token_ids, embedding_matrix, dec_embeddings, total_tokens_dec, EMBEDDING_DIM);

    // -------------------------
    // 3. Positional Encoding: Compute for a sequence length of MAX_TOKENS and add to embeddings.
    std::vector<float> pos_enc;
    cpu_positional_encoding(pos_enc, MAX_TOKENS, EMBEDDING_DIM);
    // Add positional encoding (broadcast across sentences)
    for (int i = 0; i < total_tokens_enc; i++){
        for (int j = 0; j < EMBEDDING_DIM; j++){
            enc_embeddings[i * EMBEDDING_DIM + j] += pos_enc[(i % MAX_TOKENS) * EMBEDDING_DIM + j];
        }
    }
    for (int i = 0; i < total_tokens_dec; i++){
        for (int j = 0; j < EMBEDDING_DIM; j++){
            dec_embeddings[i * EMBEDDING_DIM + j] += pos_enc[(i % MAX_TOKENS) * EMBEDDING_DIM + j];
        }
    }
    
    // -------------------------
    // 4. Initialize Encoder and Decoder Weights
    // For encoder: weights for Q, K, V, final projection (Wo), and feed-forward layers (W1, W2).
    int weight_size = d_total * d_total;
    int weight_ff_size = d_total * d_ff;
    int weight_ff2_size = d_ff * d_total;
    std::vector<float> Wq_enc(weight_size), Wk_enc(weight_size), Wv_enc(weight_size), Wo_enc(weight_size);
    std::vector<float> W1_enc(weight_ff_size), W2_enc(weight_ff2_size);
    // For decoder: similarly, for self-attention and encoder-decoder attention.
    std::vector<float> Wq_dec_self(weight_size), Wk_dec_self(weight_size),
                       Wv_dec_self(weight_size), Wo_dec_self(weight_size);
    std::vector<float> Wq_dec_encdec(weight_size), Wk_dec_encdec(weight_size),
                       Wv_dec_encdec(weight_size), Wo_dec_encdec(weight_size);
    std::vector<float> W1_dec(weight_ff_size), W2_dec(weight_ff2_size);

    // Initialize weights with a fixed pattern.
    for (int i = 0; i < weight_size; i++){
        Wq_enc[i] = static_cast<float>((i + 2) % 100) / 100.0f;
        Wk_enc[i] = static_cast<float>((i + 3) % 100) / 100.0f;
        Wv_enc[i] = static_cast<float>((i + 4) % 100) / 100.0f;
        Wo_enc[i] = static_cast<float>((i + 5) % 100) / 100.0f;
        Wq_dec_self[i] = static_cast<float>((i + 3) % 100) / 100.0f;
        Wk_dec_self[i] = static_cast<float>((i + 4) % 100) / 100.0f;
        Wv_dec_self[i] = static_cast<float>((i + 5) % 100) / 100.0f;
        Wo_dec_self[i] = static_cast<float>((i + 6) % 100) / 100.0f;
        Wq_dec_encdec[i] = static_cast<float>((i + 7) % 100) / 100.0f;
        Wk_dec_encdec[i] = static_cast<float>((i + 8) % 100) / 100.0f;
        Wv_dec_encdec[i] = static_cast<float>((i + 9) % 100) / 100.0f;
        Wo_dec_encdec[i] = static_cast<float>((i + 10) % 100) / 100.0f;
    }
    for (int i = 0; i < weight_ff_size; i++){
        W1_enc[i] = static_cast<float>((i + 6) % 100) / 100.0f;
        W1_dec[i] = static_cast<float>((i + 11) % 100) / 100.0f;
    }
    for (int i = 0; i < weight_ff2_size; i++){
        W2_enc[i] = static_cast<float>((i + 7) % 100) / 100.0f;
        W2_dec[i] = static_cast<float>((i + 12) % 100) / 100.0f;
    }

    // -------------------------
    // 5. CPU Inference
    auto cpu_start = std::chrono::high_resolution_clock::now();
    std::vector<float> enc_out_cpu;
    cpu_encoder(enc_embeddings, enc_out_cpu, Wq_enc, Wk_enc, Wv_enc, Wo_enc, W1_enc, W2_enc);
    std::vector<float> dec_out_cpu;
    cpu_decoder(dec_embeddings, enc_out_cpu, dec_out_cpu,
                Wq_dec_self, Wk_dec_self, Wv_dec_self, Wo_dec_self,
                Wq_dec_encdec, Wk_dec_encdec, Wv_dec_encdec, Wo_dec_encdec,
                W1_dec, W2_dec);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    double cpu_time = std::chrono::duration<double, std::milli>(cpu_end - cpu_start).count();
    std::cout << "CPU Total Inference Time: " << cpu_time << " ms" << std::endl;

    // -------------------------
    // 6. GPU Inference
    // For brevity the GPU implementation allocates device buffers for:
    //     - Embedding lookup and positional encoding for inputs (encoder and decoder)
    //     - Encoder weights and intermediate buffers
    //     - Decoder weights and intermediate buffers
    // Then launches kernels that mimic the CPU pipeline.
    // We time the Host-to-Device (H2D) copy time, Kernel execution, and Device-to-Host (D2H) copy time.

    hipEvent_t start_total, stop_total;
    hipEvent_t start_h2d, stop_h2d;
    hipEvent_t start_kernel, stop_kernel;
    hipEvent_t start_d2h, stop_d2h;
    CHECK_CUDA(hipEventCreate(&start_total));
    CHECK_CUDA(hipEventCreate(&stop_total));
    CHECK_CUDA(hipEventCreate(&start_h2d));
    CHECK_CUDA(hipEventCreate(&stop_h2d));
    CHECK_CUDA(hipEventCreate(&start_kernel));
    CHECK_CUDA(hipEventCreate(&stop_kernel));
    CHECK_CUDA(hipEventCreate(&start_d2h));
    CHECK_CUDA(hipEventCreate(&stop_d2h));
    CHECK_CUDA(hipEventRecord(start_total, 0));

    // (a) Copy embeddings and weights to GPU.
    float *d_enc_embed, *d_dec_embed;
    int embed_bytes = total_tokens_enc * EMBEDDING_DIM * sizeof(float);
    int dec_embed_bytes = total_tokens_dec * EMBEDDING_DIM * sizeof(float);
    CHECK_CUDA(hipMalloc(&d_enc_embed, embed_bytes));
    CHECK_CUDA(hipMalloc(&d_dec_embed, dec_embed_bytes));
    CHECK_CUDA(hipEventRecord(start_h2d, 0));
    CHECK_CUDA(hipMemcpy(d_enc_embed, enc_embeddings.data(), embed_bytes, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_dec_embed, dec_embeddings.data(), dec_embed_bytes, hipMemcpyHostToDevice));
    // (For simplicity, we assume the weights are reused on the GPU and allocate buffers for them.)
    float *d_Wq_enc, *d_Wk_enc, *d_Wv_enc, *d_Wo_enc, *d_W1_enc, *d_W2_enc;
    CHECK_CUDA(hipMalloc(&d_Wq_enc, weight_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wk_enc, weight_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wv_enc, weight_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wo_enc, weight_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_W1_enc, weight_ff_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_W2_enc, weight_ff2_size * sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_Wq_enc, Wq_enc.data(), weight_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wk_enc, Wk_enc.data(), weight_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wv_enc, Wv_enc.data(), weight_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wo_enc, Wo_enc.data(), weight_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_W1_enc, W1_enc.data(), weight_ff_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_W2_enc, W2_enc.data(), weight_ff2_size * sizeof(float), hipMemcpyHostToDevice));
    // Similarly allocate and copy decoder weights.
    float *d_Wq_dec_self, *d_Wk_dec_self, *d_Wv_dec_self, *d_Wo_dec_self;
    float *d_Wq_dec_encdec, *d_Wk_dec_encdec, *d_Wv_dec_encdec, *d_Wo_dec_encdec;
    float *d_W1_dec, *d_W2_dec;
    CHECK_CUDA(hipMalloc(&d_Wq_dec_self, weight_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wk_dec_self, weight_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wv_dec_self, weight_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wo_dec_self, weight_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wq_dec_encdec, weight_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wk_dec_encdec, weight_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wv_dec_encdec, weight_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_Wo_dec_encdec, weight_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_W1_dec, weight_ff_size * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_W2_dec, weight_ff2_size * sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_Wq_dec_self, Wq_dec_self.data(), weight_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wk_dec_self, Wk_dec_self.data(), weight_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wv_dec_self, Wv_dec_self.data(), weight_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wo_dec_self, Wo_dec_self.data(), weight_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wq_dec_encdec, Wq_dec_encdec.data(), weight_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wk_dec_encdec, Wk_dec_encdec.data(), weight_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wv_dec_encdec, Wv_dec_encdec.data(), weight_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Wo_dec_encdec, Wo_dec_encdec.data(), weight_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_W1_dec, W1_dec.data(), weight_ff_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_W2_dec, W2_dec.data(), weight_ff2_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipEventRecord(stop_h2d, 0));
    CHECK_CUDA(hipEventSynchronize(stop_h2d));
    float h2d_time;
    CHECK_CUDA(hipEventElapsedTime(&h2d_time, start_h2d, stop_h2d));

    // (b) GPU Encoder Processing
    // Allocate device buffers for encoder intermediate results.
    size_t input_size_enc = M * d_total * sizeof(float);
    size_t proj_size_enc = M * d_total * sizeof(float);
    size_t scores_size_enc = M * M * H * sizeof(float);
    size_t ffn1_size_enc = M * d_ff * sizeof(float);

    float *d_Q_enc, *d_K_enc, *d_V_enc;
    float *d_scores_enc, *d_O_enc;
    float *d_MHA_enc, *d_add1_enc, *d_ln1_enc;
    float *d_ffn1_enc, *d_ffn2_enc, *d_add2_enc, *d_ln2_enc;
    CHECK_CUDA(hipMalloc(&d_Q_enc, proj_size_enc));
    CHECK_CUDA(hipMalloc(&d_K_enc, proj_size_enc));
    CHECK_CUDA(hipMalloc(&d_V_enc, proj_size_enc));
    CHECK_CUDA(hipMalloc(&d_scores_enc, scores_size_enc));
    CHECK_CUDA(hipMalloc(&d_O_enc, proj_size_enc));
    CHECK_CUDA(hipMalloc(&d_MHA_enc, proj_size_enc));
    CHECK_CUDA(hipMalloc(&d_add1_enc, proj_size_enc));
    CHECK_CUDA(hipMalloc(&d_ln1_enc, proj_size_enc));
    CHECK_CUDA(hipMalloc(&d_ffn1_enc, ffn1_size_enc));
    CHECK_CUDA(hipMalloc(&d_ffn2_enc, proj_size_enc));
    CHECK_CUDA(hipMalloc(&d_add2_enc, proj_size_enc));
    CHECK_CUDA(hipMalloc(&d_ln2_enc, proj_size_enc));

    CHECK_CUDA(hipEventRecord(start_kernel, 0));
    // Linear Projections for encoder: Q, K, V from d_enc_embed.
    dim3 block(TILE_WIDTH, TILE_WIDTH);
    dim3 grid((d_total + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH);
    matmul_kernel<<<grid, block>>>(d_enc_embed, d_Wq_enc, d_Q_enc, M, d_total, d_total);
    matmul_kernel<<<grid, block>>>(d_enc_embed, d_Wk_enc, d_K_enc, M, d_total, d_total);
    matmul_kernel<<<grid, block>>>(d_enc_embed, d_Wv_enc, d_V_enc, M, d_total, d_total);

    // Multi-Head Attention: compute scores, softmax, weighted sum.
    dim3 grid_att((M + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH, H);
    compute_scores_kernel<<<grid_att, block>>>(d_Q_enc, d_K_enc, d_scores_enc, M, M, H, d_head);
    int threads_att = 256;
    int blocks_att = M * H;
    size_t shared_mem = threads_att * sizeof(float);
    softmax_kernel<<<blocks_att, threads_att, shared_mem>>>(d_scores_enc, M, M, H);
    dim3 grid_ws((d_head + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH, H);
    weighted_sum_kernel<<<grid_ws, block>>>(d_scores_enc, d_V_enc, d_O_enc, M, M, H, d_head);
    // Final linear projection for multi-head attention branch.
    matmul_kernel<<<grid, block>>>(d_O_enc, d_Wo_enc, d_MHA_enc, M, d_total, d_total);
    // Residual connection and layer normalization.
    int total_elems_enc = M * d_total;
    int threads_add = 256;
    int blocks_add = (total_elems_enc + threads_add - 1) / threads_add;
    add_kernel<<<blocks_add, threads_add>>>(d_enc_embed, d_MHA_enc, d_add1_enc, total_elems_enc);
    int ln_threads = 256;
    layer_norm_kernel<<<M, ln_threads, ln_threads * sizeof(float)>>>(d_add1_enc, d_ln1_enc, M, d_total, 1e-5);

    // Feed-Forward Network in encoder.
    dim3 grid_ffn((d_ff + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH);
    matmul_kernel<<<grid_ffn, block>>>(d_ln1_enc, d_W1_enc, d_ffn1_enc, M, d_total, d_ff);
    int total_ffn1 = M * d_ff;
    int blocks_relu = (total_ffn1 + threads_add - 1) / threads_add;
    relu_kernel<<<blocks_relu, threads_add>>>(d_ffn1_enc, total_ffn1);
    matmul_kernel<<<grid, block>>>(d_ffn1_enc, d_W2_enc, d_ffn2_enc, M, d_ff, d_total);
    add_kernel<<<blocks_add, threads_add>>>(d_ln1_enc, d_ffn2_enc, d_add2_enc, total_elems_enc);
    layer_norm_kernel<<<M, ln_threads, ln_threads * sizeof(float)>>>(d_add2_enc, d_ln2_enc, M, d_total, 1e-5);

    CHECK_CUDA(hipEventRecord(stop_kernel, 0));
    CHECK_CUDA(hipEventSynchronize(stop_kernel));
    float kernel_time_enc;
    CHECK_CUDA(hipEventElapsedTime(&kernel_time_enc, start_kernel, stop_kernel));

    // (c) GPU Decoder Processing (similar structure to encoder; here we process self-attention, encoder-decoder attention and FFN)
    size_t proj_size_dec = M * d_total * sizeof(float);
    size_t scores_size_dec = M * M * H * sizeof(float);
    size_t ffn1_size_dec = M * d_ff * sizeof(float);
    float *d_Q_dec, *d_K_dec, *d_V_dec;
    float *d_scores_dec, *d_O_dec;
    float *d_attn_dec;
    float *d_add_self_dec, *d_ln_self_dec;
    float *d_Q_encdec, *d_K_encdec, *d_V_encdec;
    float *d_scores_encdec, *d_O_encdec;
    float *d_attn_encdec;
    float *d_add_encdec, *d_ln_encdec;
    float *d_ffn1_dec, *d_ffn2_dec, *d_add_ffn_dec, *d_ln_dec;
    CHECK_CUDA(hipMalloc(&d_Q_dec, proj_size_dec));
    CHECK_CUDA(hipMalloc(&d_K_dec, proj_size_dec));
    CHECK_CUDA(hipMalloc(&d_V_dec, proj_size_dec));
    CHECK_CUDA(hipMalloc(&d_scores_dec, scores_size_dec));
    CHECK_CUDA(hipMalloc(&d_O_dec, proj_size_dec));
    CHECK_CUDA(hipMalloc(&d_attn_dec, proj_size_dec));
    CHECK_CUDA(hipMalloc(&d_add_self_dec, proj_size_dec));
    CHECK_CUDA(hipMalloc(&d_ln_self_dec, proj_size_dec));
    CHECK_CUDA(hipMalloc(&d_Q_encdec, proj_size_dec));
    CHECK_CUDA(hipMalloc(&d_K_encdec, proj_size_dec));
    CHECK_CUDA(hipMalloc(&d_V_encdec, proj_size_dec));
    CHECK_CUDA(hipMalloc(&d_scores_encdec, scores_size_dec));
    CHECK_CUDA(hipMalloc(&d_O_encdec, proj_size_dec));
    CHECK_CUDA(hipMalloc(&d_attn_encdec, proj_size_dec));
    CHECK_CUDA(hipMalloc(&d_add_encdec, proj_size_dec));
    CHECK_CUDA(hipMalloc(&d_ln_encdec, proj_size_dec));
    CHECK_CUDA(hipMalloc(&d_ffn1_dec, ffn1_size_dec));
    CHECK_CUDA(hipMalloc(&d_ffn2_dec, proj_size_dec));
    CHECK_CUDA(hipMalloc(&d_add_ffn_dec, proj_size_dec));
    CHECK_CUDA(hipMalloc(&d_ln_dec, proj_size_dec));

    CHECK_CUDA(hipEventRecord(start_kernel, 0));
    // Self-Attention for decoder.
    dim3 grid_dec((d_total + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH);
    matmul_kernel<<<grid_dec, block>>>(d_dec_embed, d_Wq_dec_self, d_Q_dec, M, d_total, d_total);
    matmul_kernel<<<grid_dec, block>>>(d_dec_embed, d_Wk_dec_self, d_K_dec, M, d_total, d_total);
    matmul_kernel<<<grid_dec, block>>>(d_dec_embed, d_Wv_dec_self, d_V_dec, M, d_total, d_total);
    dim3 grid_att_dec((M + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH, H);
    compute_scores_kernel<<<grid_att_dec, block>>>(d_Q_dec, d_K_dec, d_scores_dec, M, M, H, d_head);
    softmax_kernel<<<M*H, threads_att, shared_mem>>>(d_scores_dec, M, M, H);
    weighted_sum_kernel<<<grid_att_dec, block>>>(d_scores_dec, d_V_dec, d_O_dec, M, M, H, d_head);
    matmul_kernel<<<grid_dec, block>>>(d_O_dec, d_Wo_dec_self, d_attn_dec, M, d_total, d_total);
    add_kernel<<<blocks_add, threads_add>>>(d_dec_embed, d_attn_dec, d_add_self_dec, M * d_total);
    layer_norm_kernel<<<M, ln_threads, ln_threads * sizeof(float)>>>(d_add_self_dec, d_ln_self_dec, M, d_total, 1e-5);
    
    // Encoder-Decoder Attention:
    matmul_kernel<<<grid_dec, block>>>(d_ln_self_dec, d_Wq_dec_encdec, d_Q_encdec, M, d_total, d_total);
    matmul_kernel<<<grid_dec, block>>>(d_enc, d_Wk_dec_encdec, d_K_encdec, M, d_total, d_total);
    matmul_kernel<<<grid_dec, block>>>(d_enc, d_Wv_dec_encdec, d_V_encdec, M, d_total, d_total);
    dim3 grid_att_encdec((M + TILE_WIDTH - 1) / TILE_WIDTH, (M + TILE_WIDTH - 1) / TILE_WIDTH, H);
    compute_scores_kernel<<<grid_att_encdec, block>>>(d_Q_encdec, d_K_encdec, d_scores_encdec, M, M, H, d_head);
    softmax_kernel<<<M*H, threads_att, shared_mem>>>(d_scores_encdec, M, M, H);
    weighted_sum_kernel<<<grid_att_encdec, block>>>(d_scores_encdec, d_V_encdec, d_O_encdec, M, M, H, d_head);
    matmul_kernel<<<grid_dec, block>>>(d_O_encdec, d_Wo_dec_encdec, d_attn_encdec, M, d_total, d_total);
    add_kernel<<<blocks_add, threads_add>>>(d_ln_self_dec, d_attn_encdec, d_add_encdec, M * d_total);
    layer_norm_kernel<<<M, ln_threads, ln_threads * sizeof(float)>>>(d_add_encdec, d_ln_encdec, M, d_total, 1e-5);
    
    // Feed-Forward Network in decoder.
    matmul_kernel<<<grid_ffn, block>>>(d_ln_encdec, d_W1_dec, d_ffn1_dec, M, d_total, d_ff);
    int total_ffn1_dec = M * d_ff;
    int blocks_relu_dec = (total_ffn1_dec + threads_add - 1) / threads_add;
    relu_kernel<<<blocks_relu_dec, threads_add>>>(d_ffn1_dec, total_ffn1_dec);
    matmul_kernel<<<grid_dec, block>>>(d_ffn1_dec, d_W2_dec, d_ffn2_dec, M, d_ff, d_total);
    add_kernel<<<blocks_add, threads_add>>>(d_ln_encdec, d_ffn2_dec, d_add_ffn_dec, M * d_total);
    layer_norm_kernel<<<M, ln_threads, ln_threads * sizeof(float)>>>(d_add_ffn_dec, d_ln_dec, M, d_total, 1e-5);
    
    CHECK_CUDA(hipEventRecord(stop_kernel, 0));
    CHECK_CUDA(hipEventSynchronize(stop_kernel));
    float kernel_time_dec;
    CHECK_CUDA(hipEventElapsedTime(&kernel_time_dec, start_kernel, stop_kernel));

    // (d) Copy final decoder output from device to host.
    std::vector<float> dec_out_gpu(M * d_total);
    CHECK_CUDA(hipEventRecord(start_d2h, 0));
    CHECK_CUDA(hipMemcpy(dec_out_gpu.data(), d_ln_dec, proj_size_dec, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipEventRecord(stop_d2h, 0));
    CHECK_CUDA(hipEventSynchronize(stop_d2h));
    float d2h_time;
    CHECK_CUDA(hipEventElapsedTime(&d2h_time, start_d2h, stop_d2h));
    
    CHECK_CUDA(hipEventRecord(stop_total, 0));
    CHECK_CUDA(hipEventSynchronize(stop_total));
    float total_gpu_time;
    CHECK_CUDA(hipEventElapsedTime(&total_gpu_time, start_total, stop_total));

    std::cout << "GPU Inference Timings:" << std::endl;
    std::cout << "  H2D Copy Time: " << h2d_time << " ms" << std::endl;
    std::cout << "  Encoder Kernel Time: " << kernel_time_enc << " ms" << std::endl;
    std::cout << "  Decoder Kernel Time: " << kernel_time_dec << " ms" << std::endl;
    std::cout << "  D2H Copy Time: " << d2h_time << " ms" << std::endl;
    std::cout << "  Total GPU Inference Time: " << total_gpu_time << " ms" << std::endl;

    // ---------------------------------------------------------------------
    // (Optional) Compare or print some outputs...
    std::cout << "CPU Decoder Output (first 10 values):" << std::endl;
    for (int i = 0; i < 10; i++){
        std::cout << dec_out_cpu[i] << " ";
    }
    std::cout << "\nGPU Decoder Output (first 10 values):" << std::endl;
    for (int i = 0; i < 10; i++){
        std::cout << dec_out_gpu[i] << " ";
    }
    std::cout << std::endl;

    // ---------------------------------------------------------------------
    // Cleanup CPU memory
    // (Vectors allocated on the host will free automatically)
    // Cleanup GPU memory and events.
    hipFree(d_enc_embed);  hipFree(d_dec_embed);
    hipFree(d_Wq_enc);  hipFree(d_Wk_enc);  hipFree(d_Wv_enc);  hipFree(d_Wo_enc);
    hipFree(d_W1_enc);  hipFree(d_W2_enc);
    hipFree(d_Wq_dec_self); hipFree(d_Wk_dec_self); hipFree(d_Wv_dec_self); hipFree(d_Wo_dec_self);
    hipFree(d_Wq_dec_encdec); hipFree(d_Wk_dec_encdec); hipFree(d_Wv_dec_encdec); hipFree(d_Wo_dec_encdec);
    hipFree(d_W1_dec);  hipFree(d_W2_dec);
    hipFree(d_Q_enc);  hipFree(d_K_enc);  hipFree(d_V_enc);
    hipFree(d_scores_enc);  hipFree(d_O_enc);
    hipFree(d_MHA_enc);  hipFree(d_add1_enc);  hipFree(d_ln1_enc);
    hipFree(d_ffn1_enc); hipFree(d_ffn2_enc);
    hipFree(d_add2_enc); hipFree(d_ln2_enc);
    hipFree(d_Q_dec);  hipFree(d_K_dec);  hipFree(d_V_dec);
    hipFree(d_scores_dec);  hipFree(d_O_dec);
    hipFree(d_attn_dec);  hipFree(d_add_self_dec);  hipFree(d_ln_self_dec);
    hipFree(d_Q_encdec); hipFree(d_K_encdec); hipFree(d_V_encdec);
    hipFree(d_scores_encdec); hipFree(d_O_encdec);
    hipFree(d_attn_encdec);  hipFree(d_add_encdec); hipFree(d_ln_encdec);
    hipFree(d_ffn1_dec); hipFree(d_ffn2_dec);
    hipFree(d_add_ffn_dec); hipFree(d_ln_dec);
    hipEventDestroy(start_total); hipEventDestroy(stop_total);
    hipEventDestroy(start_h2d); hipEventDestroy(stop_h2d);
    hipEventDestroy(start_kernel); hipEventDestroy(stop_kernel);
    hipEventDestroy(start_d2h); hipEventDestroy(stop_d2h);

    return 0;
}

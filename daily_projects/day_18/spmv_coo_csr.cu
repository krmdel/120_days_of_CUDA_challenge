#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Error checking macro
#define CUDA_CHECK(call) {                                               \
    hipError_t err = call;                                              \
    if(err != hipSuccess) {                                             \
        fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__,  \
                hipGetErrorString(err));                                \
        exit(err);                                                       \
    }                                                                    \
}

__global__ void spmv_coo_kernel(int nnz, const int *cooRow, const int *cooCol, 
                                const float *cooVal, const float *x, float *y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nnz) {
        // Each thread computes its product and atomically adds to the correct output element.
        atomicAdd(&y[cooRow[idx]], cooVal[idx] * x[cooCol[idx]]);
    }
}

__global__ void spmv_csr_kernel(int num_rows, const int *csrRowPtr, const int *csrCol, 
                                const float *csrVal, const float *x, float *y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < num_rows) {
        float dot = 0.0f;
        int row_start = csrRowPtr[row];
        int row_end   = csrRowPtr[row+1];
        for (int i = row_start; i < row_end; i++) {
            dot += csrVal[i] * x[csrCol[i]];
        }
        y[row] = dot;
    }
}

int main() {
    const int num_rows = 4;
    const int num_cols = 4;
    const int nnz = 6;

    int h_cooRow[] = {0, 0, 1, 2, 3, 3};
    int h_cooCol[] = {0, 1, 1, 2, 0, 3};
    float h_cooVal[] = {10.0f, 20.0f, 30.0f, 40.0f, 50.0f, 60.0f};

    int h_csrRowPtr[] = {0, 2, 3, 4, 6}; // row i goes from h_csrRowPtr[i] to h_csrRowPtr[i+1]-1
    int h_csrCol[] = {0, 1, 1, 2, 0, 3};
    float h_csrVal[] = {10.0f, 20.0f, 30.0f, 40.0f, 50.0f, 60.0f};

    float h_x[] = {1.0f, 2.0f, 3.0f, 4.0f};
    float h_y_coo[4] = {0};
    float h_y_csr[4] = {0};

    hipEvent_t start, stop;
    float elapsedTime = 0.0f;

    int *d_cooRow, *d_cooCol;
    float *d_cooVal, *d_x, *d_y;
    CUDA_CHECK(hipMalloc((void**)&d_cooRow, nnz * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_cooCol, nnz * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_cooVal, nnz * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_x, num_cols * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_y, num_rows * sizeof(float)));

    CUDA_CHECK(hipMemset(d_y, 0, num_rows * sizeof(float)));

    // Copy data from CPU to GPU
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    hipEventRecord(start, 0);
    CUDA_CHECK(hipMemcpy(d_cooRow, h_cooRow, nnz * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_cooCol, h_cooCol, nnz * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_cooVal, h_cooVal, nnz * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_x, h_x, num_cols * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("COO copy time: %f ms\n", elapsedTime);
    float cooHtoD = elapsedTime;

    // Launch COO kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (nnz + threadsPerBlock - 1) / threadsPerBlock;
    hipEventRecord(start, 0);
    spmv_coo_kernel<<<blocksPerGrid, threadsPerBlock>>>(nnz, d_cooRow, d_cooCol, d_cooVal, d_x, d_y);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("COO kernel time: %f ms\n", elapsedTime);
    float cooKernel = elapsedTime;

    // Copy data back to CPU
    hipEventRecord(start, 0);
    CUDA_CHECK(hipMemcpy(h_y_coo, d_y, num_rows * sizeof(float), hipMemcpyDeviceToHost));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("COO copy time: %f ms\n", elapsedTime);
    float cooDtoH = elapsedTime;

    float cooTotal = cooHtoD + cooKernel + cooDtoH;
    printf("COO total time: %f ms\n\n", cooTotal);

    CUDA_CHECK(hipFree(d_cooRow));
    CUDA_CHECK(hipFree(d_cooCol));
    CUDA_CHECK(hipFree(d_cooVal));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));

    int *d_csrRowPtr, *d_csrCol;
    float *d_csrVal, *d_x2, *d_y2;
    CUDA_CHECK(hipMalloc((void**)&d_csrRowPtr, (num_rows + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_csrCol, nnz * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_csrVal, nnz * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_x2, num_cols * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_y2, num_rows * sizeof(float)));

    CUDA_CHECK(hipMemset(d_y2, 0, num_rows * sizeof(float)));

    // Copy data from CPU to GPU
    hipEventRecord(start, 0);
    CUDA_CHECK(hipMemcpy(d_csrRowPtr, h_csrRowPtr, (num_rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_csrCol, h_csrCol, nnz * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_csrVal, h_csrVal, nnz * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_x2, h_x, num_cols * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("CSR copy time: %f ms\n", elapsedTime);
    float csrHtoD = elapsedTime;

    // Launch CSR kernel
    threadsPerBlock = 256;
    blocksPerGrid = (num_rows + threadsPerBlock - 1) / threadsPerBlock;
    hipEventRecord(start, 0);
    spmv_csr_kernel<<<blocksPerGrid, threadsPerBlock>>>(num_rows, d_csrRowPtr, d_csrCol, d_csrVal, d_x2, d_y2);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("CSR kernel time: %f ms\n", elapsedTime);
    float csrKernel = elapsedTime;

    // Copy data back to CPU
    hipEventRecord(start, 0);
    CUDA_CHECK(hipMemcpy(h_y_csr, d_y2, num_rows * sizeof(float), hipMemcpyDeviceToHost));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("CSR copy time: %f ms\n", elapsedTime);
    float csrDtoH = elapsedTime;

    float csrTotal = csrHtoD + csrKernel + csrDtoH;
    printf("CSR total time: %f ms\n\n", csrTotal);

    // Clean up
    CUDA_CHECK(hipFree(d_csrRowPtr));
    CUDA_CHECK(hipFree(d_csrCol));
    CUDA_CHECK(hipFree(d_csrVal));
    CUDA_CHECK(hipFree(d_x2));
    CUDA_CHECK(hipFree(d_y2));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return 0;
}

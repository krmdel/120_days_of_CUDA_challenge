#include <chrono>
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

__host__ __device__ float f(float x, float y) {
    return x + y;
}

void vecadd_cpu(float* x, float* y, float* z, int N) {
    for (unsigned int i = 0; i < N; i++) {
        z[i] = f(x[i], y[i]);
    }
}

__global__ void vecadd_kernel(float* x, float* y, float* z, int N) {
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        z[i] = f(x[i], y[i]);
    }
}

void vecadd_gpu(float* x, float* y, float* z, int N) {
    // Allocate memory on GPU
    float *x_d, *y_d, *z_d;
    hipMalloc((void**)&x_d, N * sizeof(float));
    hipMalloc((void**)&y_d, N * sizeof(float));
    hipMalloc((void**)&z_d, N * sizeof(float));

    // Copy data to GPU
    hipMemcpy(x_d, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, N * sizeof(float), hipMemcpyHostToDevice);

    // Setup kernel launch parameters
    const unsigned int numThreadsPerBlock = 512;
    const unsigned int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;

    // Create CUDA events for kernel timing
    hipEvent_t kernel_start, kernel_stop;
    hipEventCreate(&kernel_start);
    hipEventCreate(&kernel_stop);

    // Record kernel start time, launch kernel, then record stop time
    hipEventRecord(kernel_start);
    vecadd_kernel<<< numBlocks, numThreadsPerBlock >>>(x_d, y_d, z_d, N);
    hipError_t err = hipDeviceSynchronize(); // Wait for the kernel to finish to execute the timing
    if (err != hipSuccess) {
        std::cerr << "CUDA error in file '" << __FILE__ << "' in line " << __LINE__ << ": " << hipGetErrorString(err) << "\n";
        exit(EXIT_FAILURE);
    }
    hipEventRecord(kernel_stop);

    // Wait for the kernel to finish
    hipEventSynchronize(kernel_stop);

    // Calculate and print kernel elapsed time
    float kernel_time_ms = 0;
    hipEventElapsedTime(&kernel_time_ms, kernel_start, kernel_stop);
    std::cout << "GPU kernel elapsed time: " << kernel_time_ms << " ms\n";

    // Copy result back to CPU
    hipMemcpy(z, z_d, N * sizeof(float), hipMemcpyDeviceToHost);

    // Clean up GPU memory and CUDA events for kernel timing
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
    hipEventDestroy(kernel_start);
    hipEventDestroy(kernel_stop);
}

int main(int argc, char** argv) {
    hipDeviceSynchronize();

    // Allocate memory on host
    unsigned int N = (argc > 1) ? atoi(argv[1]) : (1 << 25);
    float* x = (float*)malloc(N * sizeof(float));
    float* y = (float*)malloc(N * sizeof(float));
    float* z = (float*)malloc(N * sizeof(float));
    for (unsigned int i = 0; i < N; i++) {
        x[i] = rand();
        y[i] = rand();
    }

    // CPU timing using chrono
    auto start_cpu = std::chrono::high_resolution_clock::now();
    vecadd_cpu(x, y, z, N);
    auto end_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpu_duration = end_cpu - start_cpu;
    std::cout << "CPU vecadd elapsed time: " << cpu_duration.count() * 1000 << " ms\n";

    // Overall GPU timing (including memory transfers) using CUDA events
    hipEvent_t overall_start, overall_stop;
    hipEventCreate(&overall_start);
    hipEventCreate(&overall_stop);

    hipEventRecord(overall_start);
    vecadd_gpu(x, y, z, N);
    hipEventRecord(overall_stop);

    hipEventSynchronize(overall_stop);
    float overall_gpu_time_ms = 0;
    hipEventElapsedTime(&overall_gpu_time_ms, overall_start, overall_stop);
    std::cout << "Overall GPU vecadd elapsed time: " << overall_gpu_time_ms << " ms\n";

    // Clean up host memory and overall timing events
    free(x);
    free(y);
    free(z);
    hipEventDestroy(overall_start);
    hipEventDestroy(overall_stop);

    return 0;
}

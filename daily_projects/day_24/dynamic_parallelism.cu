#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void childKernel(int *d_data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        d_data[idx] *= 2;
    }
}

__global__ void parentKernel(int *d_data, int n) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        int threadsPerBlock = 256;
        int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;
        childKernel<<<blocks, threadsPerBlock>>>(d_data, n);
    }
}

int main(void) {
    int n = 1 << 20; // 1M elements
    size_t size = n * sizeof(int);

    // Allocate and initialize host memory
    int *h_data = (int*) malloc(size);
    for (int i = 0; i < n; i++) {
        h_data[i] = i;
    }

    // Allocate device memory
    int *d_data;
    hipMalloc((void**)&d_data, size);

    hipEvent_t startTotal, stopTotal, start, stop;
    hipEventCreate(&startTotal);
    hipEventCreate(&stopTotal);
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(startTotal, 0);

    // Host to Device copy timing
    hipEventRecord(start, 0);
    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float timeHTD;
    hipEventElapsedTime(&timeHTD, start, stop);

    // Kernel execution timing (parent and dynamic child kernel)
    hipEventRecord(start, 0);
    // Launch parent kernel with 1 block and 32 threads
    parentKernel<<<1, 32>>>(d_data, n);
    // Synchronize on the host to wait for the parent kernel and all child kernels
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float timeKernel;
    hipEventElapsedTime(&timeKernel, start, stop);

    // Device to Host copy timing
    hipEventRecord(start, 0);
    hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float timeDTH;
    hipEventElapsedTime(&timeDTH, start, stop);

    hipEventRecord(stopTotal, 0);
    hipEventSynchronize(stopTotal);
    float totalTime;
    hipEventElapsedTime(&totalTime, startTotal, stopTotal);

    printf("Time for CPU to GPU copy: %f ms\n", timeHTD);
    printf("Time for kernel execution: %f ms\n", timeKernel);
    printf("Time for GPU to CPU copy: %f ms\n", timeDTH);
    printf("Total time: %f ms\n", totalTime);

    // Clean up
    hipEventDestroy(startTotal);
    hipEventDestroy(stopTotal);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_data);
    free(h_data);

    return 0;
}

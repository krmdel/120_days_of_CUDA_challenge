#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>


#include <chrono>
#include <cmath>
#include <complex>
#include <iomanip>
#include <iostream>
#include <random>
#include <vector>

#define CUDA_CHECK(x)                                                         \
    do {                                                                      \
        hipError_t rc = (x);                                                 \
        if (rc != hipSuccess) {                                              \
            std::cerr << "CUDA error: " << hipGetErrorString(rc)             \
                      << " (" << __FILE__ << ":" << __LINE__ << ")\n";        \
            std::exit(EXIT_FAILURE);                                          \
        }                                                                     \
    } while (0)

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

// GPU kernel: each thread computes one output coefficient k
__global__ void dft1d_kernel(const float* __restrict__ x,
                             hipFloatComplex* __restrict__ X,
                             int N) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k >= N) return;

    float real = 0.f, imag = 0.f;
    const float twopi_over_N = -2.f * M_PI / N;

    for (int n = 0; n < N; ++n) {
        float angle = twopi_over_N * k * n;
        float s, c;
        sincosf(angle, &s, &c);
        float val = x[n];
        real += val * c;
        imag += val * s;
    }
    X[k] = make_hipFloatComplex(real, imag);
}

// CPU reference (single-thread, baseline)
void dft1d_cpu(const std::vector<float>& in,
               std::vector<std::complex<float>>& out) {
    const int N = static_cast<int>(in.size());
    out.resize(N);
    const float twopi_over_N = -2.f * M_PI / N;

    for (int k = 0; k < N; ++k) {
        float real = 0.f, imag = 0.f;
        for (int n = 0; n < N; ++n) {
            float angle = twopi_over_N * k * n;
            real += in[n] * std::cos(angle);
            imag += in[n] * std::sin(angle);
        }
        out[k] = {real, imag};
    }
}

int main(int argc, char** argv) {
    const int N = (argc > 1) ? std::atoi(argv[1]) : 16384; // 128 x 128
    std::cout << "1-D signal length: " << N << "\n\n";

    // Generate random real signal
    std::vector<float> h_signal(N);
    std::mt19937 rng(123);
    std::uniform_real_distribution<float> dist(0.f, 1.f);
    for (float& v : h_signal) v = dist(rng);

    // CPU baseline
    std::vector<std::complex<float>> h_dft_cpu;
    auto t0 = std::chrono::high_resolution_clock::now();
    dft1d_cpu(h_signal, h_dft_cpu);
    auto t1 = std::chrono::high_resolution_clock::now();
    double cpu_ms =
        std::chrono::duration<double, std::milli>(t1 - t0).count();
    std::cout << "CPU DFT     : " << std::fixed << std::setprecision(3)
              << cpu_ms << " ms\n";

    // Allocate GPU buffers
    float*            d_signal = nullptr;
    hipFloatComplex*   d_dft    = nullptr;
    CUDA_CHECK(hipMalloc(&d_signal, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_dft,    N * sizeof(hipFloatComplex)));

    // Events for fine-grained timing
    hipEvent_t eH2D0, eH2D1, eK0, eK1, eD2H0, eD2H1;
    hipEventCreate(&eH2D0); hipEventCreate(&eH2D1);
    hipEventCreate(&eK0);   hipEventCreate(&eK1);
    hipEventCreate(&eD2H0); hipEventCreate(&eD2H1);

    // H2D copy
    hipEventRecord(eH2D0);
    CUDA_CHECK(hipMemcpy(d_signal, h_signal.data(),
                          N * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(eH2D1);

    // Launch kernel (1-D grid)
    const int THREADS = 256;
    const int BLOCKS  = (N + THREADS - 1) / THREADS;

    hipEventRecord(eK0);
    dft1d_kernel<<<BLOCKS, THREADS>>>(d_signal, d_dft, N);
    hipEventRecord(eK1);

    // Copy result back
    std::vector<hipFloatComplex> h_dft_gpu(N);
    hipEventRecord(eD2H0);
    CUDA_CHECK(hipMemcpy(h_dft_gpu.data(), d_dft,
                          N * sizeof(hipFloatComplex),
                          hipMemcpyDeviceToHost));
    hipEventRecord(eD2H1);

    CUDA_CHECK(hipDeviceSynchronize());

    // GPU timings
    float h2d_ms, k_ms, d2h_ms;
    hipEventElapsedTime(&h2d_ms, eH2D0, eH2D1);
    hipEventElapsedTime(&k_ms,   eK0,   eK1);
    hipEventElapsedTime(&d2h_ms, eD2H0, eD2H1);

    std::cout << "GPU H2D     : " << h2d_ms << " ms\n";
    std::cout << "GPU Kernel  : " << k_ms   << " ms\n";
    std::cout << "GPU D2H     : " << d2h_ms << " ms\n";
    std::cout << "GPU Total   : " << (h2d_ms + k_ms + d2h_ms) << " ms\n";

    // Cleanup
    hipFree(d_signal);
    hipFree(d_dft);
    hipEventDestroy(eH2D0); hipEventDestroy(eH2D1);
    hipEventDestroy(eK0);   hipEventDestroy(eK1);
    hipEventDestroy(eD2H0); hipEventDestroy(eD2H1);

    return 0;
}

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Kernel for matrix multiplication:
// A is N x M, B is M x K, and C is N x K.
__global__ void matmul_kernel(float* A, float* B, float* C,
                              unsigned int N, unsigned int M, unsigned int K) {
    // Compute the row index for the output matrix C
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    // Compute the column index for the output matrix C
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundary conditions for the output matrix
    if (row < N && col < K) {
        float sum = 0.0f;
        // Accumulate over the shared dimension M
        for (unsigned int i = 0; i < M; i++) {
            // A is stored in row-major order: element at (row, i)
            // B is stored in row-major order: element at (i, col)
            sum += A[row * M + i] * B[i * K + col];
        }
        // Write the result to C at (row, col)
        C[row * K + col] = sum;
    }
}

void matmul_gpu(float* A, float* B, float* C,
                unsigned int N, unsigned int M, unsigned int K) {
    // Create CUDA events for timing different stages
    hipEvent_t start_alloc, stop_alloc;
    hipEvent_t start_h2d, stop_h2d;
    hipEvent_t start_kernel, stop_kernel;
    hipEvent_t start_d2h, stop_d2h;
    float time_alloc, time_h2d, time_kernel, time_d2h;

    hipEventCreate(&start_alloc);
    hipEventCreate(&stop_alloc);
    hipEventCreate(&start_h2d);
    hipEventCreate(&stop_h2d);
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventCreate(&start_d2h);
    hipEventCreate(&stop_d2h);

    float *A_d, *B_d, *C_d;

    // Allocate memory on GPU
    hipEventRecord(start_alloc, 0);
    hipMalloc((void**)&A_d, N * M * sizeof(float));
    hipMalloc((void**)&B_d, M * K * sizeof(float));
    hipMalloc((void**)&C_d, N * K * sizeof(float));
    hipDeviceSynchronize();
    hipEventRecord(stop_alloc, 0);
    hipEventSynchronize(stop_alloc);
    hipEventElapsedTime(&time_alloc, start_alloc, stop_alloc);
    printf("Time for memory allocation: %f ms\n", time_alloc);

    // Copy data from CPU to GPU
    hipEventRecord(start_h2d, 0);
    hipMemcpy(A_d, A, N * M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    hipEventRecord(stop_h2d, 0);
    hipEventSynchronize(stop_h2d);
    hipEventElapsedTime(&time_h2d, start_h2d, stop_h2d);
    printf("Time for host-to-device copy: %f ms\n", time_h2d);

    // Launch the kernel
    hipEventRecord(start_kernel, 0);
    // Define a 2D grid for the output matrix which is N x K
    dim3 numThreadsPerBlock(32, 32);
    dim3 numBlocks((K + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x,
                   (N + numThreadsPerBlock.y - 1) / numThreadsPerBlock.y);
    matmul_kernel<<<numBlocks, numThreadsPerBlock>>>(A_d, B_d, C_d, N, M, K);
    hipDeviceSynchronize();
    hipEventRecord(stop_kernel, 0);
    hipEventSynchronize(stop_kernel);
    hipEventElapsedTime(&time_kernel, start_kernel, stop_kernel);
    printf("Time for kernel execution: %f ms\n", time_kernel);

    // Copy the result from GPU to CPU
    hipEventRecord(start_d2h, 0);
    hipMemcpy(C, C_d, N * K * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop_d2h, 0);
    hipEventSynchronize(stop_d2h);
    hipEventElapsedTime(&time_d2h, start_d2h, stop_d2h);
    printf("Time for device-to-host copy: %f ms\n", time_d2h);

    // Free GPU memory
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    hipDeviceSynchronize();

    // Clean up events
    hipEventDestroy(start_alloc);
    hipEventDestroy(stop_alloc);
    hipEventDestroy(start_h2d);
    hipEventDestroy(stop_h2d);
    hipEventDestroy(start_kernel);
    hipEventDestroy(stop_kernel);
    hipEventDestroy(start_d2h);
    hipEventDestroy(stop_d2h);
}

int main() {
    // Dimensions for the multiplication: A is N x M, B is M x K, C is N x K.
    unsigned int N = 1024; // Number of rows in A and C
    unsigned int M = 512;  // Number of columns in A and rows in B
    unsigned int K = 768;  // Number of columns in B and C

    // Allocate memory on CPU for matrices A, B, and C
    float *A = (float*)malloc(N * M * sizeof(float));
    float *B = (float*)malloc(M * K * sizeof(float));
    float *C = (float*)malloc(N * K * sizeof(float));

    // Initialize matrices A and B with sample values
    for (unsigned int i = 0; i < N * M; i++) {
        A[i] = 1.0f; // For example, fill A with 1's.
    }
    for (unsigned int i = 0; i < M * K; i++) {
        B[i] = 2.0f; // For example, fill B with 2's.
    }

    // Call the function to perform matrix multiplication
    matmul_gpu(A, B, C, N, M, K);

    // Optionally, you could print some elements of C here for verification.

    // Free CPU memory
    free(A);
    free(B);
    free(C);

    return 0;
}

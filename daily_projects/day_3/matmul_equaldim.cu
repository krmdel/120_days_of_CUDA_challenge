#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void matmul_kernel(float* a, float* b, float* c, unsigned int N) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.0f;
        for (unsigned int i = 0; i < N; i++) {
            sum += a[row * N + i] * b[i * N + col];
        }
        c[row * N + col] = sum;
    }
}

void matmul_gpu(float* a, float* b, float* c, unsigned int N) {
    hipEvent_t start_alloc, stop_alloc;
    hipEvent_t start_h2d, stop_h2d;
    hipEvent_t start_kernel, stop_kernel;
    hipEvent_t start_d2h, stop_d2h;
    float time_alloc, time_h2d, time_kernel, time_d2h;

    hipEventCreate(&start_alloc);
    hipEventCreate(&stop_alloc);
    hipEventCreate(&start_h2d);
    hipEventCreate(&stop_h2d);
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventCreate(&start_d2h);
    hipEventCreate(&stop_d2h);

    float *a_d, *b_d, *c_d;

    // Allocate memory on GPU
    hipEventRecord(start_alloc, 0);
    hipMalloc((void**)&a_d, N * N * sizeof(float));
    hipMalloc((void**)&b_d, N * N * sizeof(float));
    hipMalloc((void**)&c_d, N * N * sizeof(float));
    hipDeviceSynchronize();
    hipEventRecord(stop_alloc, 0);
    hipEventSynchronize(stop_alloc);
    hipEventElapsedTime(&time_alloc, start_alloc, stop_alloc);
    printf("Time for memory allocation: %f ms\n", time_alloc);

    // Copy data from CPU to GPU
    hipEventRecord(start_h2d, 0);
    hipMemcpy(a_d, a, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    hipEventRecord(stop_h2d, 0);
    hipEventSynchronize(stop_h2d);
    hipEventElapsedTime(&time_h2d, start_h2d, stop_h2d);
    printf("Time for host-to-device copy: %f ms\n", time_h2d);

    // Calling the kernel and performing the operation
    hipEventRecord(start_kernel, 0);
    dim3 numThreadsPerBlock(32, 32);
    dim3 numBlocks((N + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x,
                   (N + numThreadsPerBlock.y - 1) / numThreadsPerBlock.y);
    matmul_kernel<<<numBlocks, numThreadsPerBlock>>>(a_d, b_d, c_d, N);
    hipDeviceSynchronize();
    hipEventRecord(stop_kernel, 0);
    hipEventSynchronize(stop_kernel);
    hipEventElapsedTime(&time_kernel, start_kernel, stop_kernel);
    printf("Time for kernel execution: %f ms\n", time_kernel);

    // Copy the result from GPU to CPU
    hipEventRecord(start_d2h, 0);
    hipMemcpy(c, c_d, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop_d2h, 0);
    hipEventSynchronize(stop_d2h);
    hipEventElapsedTime(&time_d2h, start_d2h, stop_d2h);
    printf("Time for device-to-host copy: %f ms\n", time_d2h);

    // Free GPU memory
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    hipDeviceSynchronize();

    // Clean up events
    hipEventDestroy(start_alloc);
    hipEventDestroy(stop_alloc);
    hipEventDestroy(start_h2d);
    hipEventDestroy(stop_h2d);
    hipEventDestroy(start_kernel);
    hipEventDestroy(stop_kernel);
    hipEventDestroy(start_d2h);
    hipEventDestroy(stop_d2h);
}

int main() {
    unsigned int N = 1024; // Dimension of the square matrices.

    // Allocate memory on CPU for matrices A, B, and C
    float *a = (float*)malloc(N * N * sizeof(float));
    float *b = (float*)malloc(N * N * sizeof(float));
    float *c = (float*)malloc(N * N * sizeof(float));

    // Initialize matrices A and B with sample values
    for (int i = 0; i < N * N; i++) {
        a[i] = 1.0f; // For example, fill A with 1's.
        b[i] = 2.0f; // For example, fill B with 2's.
    }

    // Call the function to perform matrix multiplication
    matmul_gpu(a, b, c, N);

    // Free CPU memory
    free(a);
    free(b);
    free(c);

    return 0;
}

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#include <hipfft/hipfft.h>

#include <chrono>
#include <cmath>
#include <complex>
#include <iomanip>
#include <iostream>
#include <random>
#include <vector>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

// Error macros
#define CUDA_CHECK(call)                                                     \
    do {                                                                     \
        hipError_t err = call;                                              \
        if (err != hipSuccess) {                                            \
            std::cerr << "CUDA error " << hipGetErrorString(err)            \
                      << " (" << __FILE__ << ":" << __LINE__ << ")\n";       \
            std::exit(EXIT_FAILURE);                                         \
        }                                                                    \
    } while (0)

#define CUFFT_CHECK(call)                                                    \
    do {                                                                     \
        hipfftResult err = call;                                              \
        if (err != HIPFFT_SUCCESS) {                                          \
            std::cerr << "cuFFT error " << err                               \
                      << " (" << __FILE__ << ":" << __LINE__ << ")\n";       \
            std::exit(EXIT_FAILURE);                                         \
        }                                                                    \
    } while (0)

// Incremental rotation helper
__device__ __forceinline__ void rotate(float& c, float& s,
                                       const float cD, const float sD)
{
    const float t = c * cD - s * sD;
    s             = c * sD + s * cD;
    c             = t;
}

// Tiled kernel
constexpr int TILE = 16;

__global__ void dft2d_kernel_tiled(const float* __restrict__ src,
                                   hipFloatComplex* __restrict__ dst,
                                   int W, int H)
{
    const int u = blockIdx.x * blockDim.x + threadIdx.x;
    const int v = blockIdx.y * blockDim.y + threadIdx.y;
    if (u >= W || v >= H) return;

    const float kx = -2.f * M_PI * static_cast<float>(u) / W;
    const float ky = -2.f * M_PI * static_cast<float>(v) / H;
    float s_dx, c_dx, s_dy, c_dy;
    sincosf(kx, &s_dx, &c_dx);
    sincosf(ky, &s_dy, &c_dy);

    __shared__ float tile[TILE][TILE];

    float accRe = 0.f, accIm = 0.f;

    for (int y0 = 0; y0 < H; y0 += TILE) {
        float s_yRow, c_yRow;
        sincosf(ky * static_cast<float>(y0), &s_yRow, &c_yRow);

        for (int x0 = 0; x0 < W; x0 += TILE) {
            // Load tile to shared memory
            int gX = x0 + threadIdx.x;
            int gY = y0 + threadIdx.y;
            tile[threadIdx.y][threadIdx.x] =
                (gX < W && gY < H) ? src[gY * W + gX] : 0.f;
            __syncthreads();

            float s_xCol, c_xCol;
            sincosf(kx * static_cast<float>(x0), &s_xCol, &c_xCol);

            float s_y = s_yRow, c_y = c_yRow;
            #pragma unroll
            for (int ty = 0; ty < TILE && (y0 + ty) < H; ++ty) {
                float s_x = s_xCol, c_x = c_xCol;

                #pragma unroll
                for (int tx = 0; tx < TILE && (x0 + tx) < W; ++tx) {
                    const float cosTot = c_x * c_y - s_x * s_y;
                    const float sinTot = c_x * s_y + s_x * c_y;
                    const float val    = tile[ty][tx];
                    accRe += val * cosTot;
                    accIm += val * sinTot;

                    rotate(c_x, s_x, c_dx, s_dx); // +1 pixel in x
                }
                rotate(c_y, s_y, c_dy, s_dy);     // +1 pixel in y
            }
            __syncthreads();
        }
    }
    dst[v * W + u] = make_hipFloatComplex(accRe, accIm);
}

int main(int argc, char** argv)
{
    const int W = (argc > 1) ? std::atoi(argv[1]) : 128;
    const int H = (argc > 2) ? std::atoi(argv[2]) : 128;
    const size_t N = static_cast<size_t>(W) * H;

    std::cout << "Image " << W << "×" << H
              << " (" << N << " px)\n\n";

    // Generate random image (real)
    std::vector<float> h_img(N);
    std::mt19937 rng(42);
    std::uniform_real_distribution<float> dist(0.f, 1.f);
    for (float& p : h_img) p = dist(rng);

    
    // Buffer Allocation
    float*           d_img      = nullptr;            // input for tiled kernel
    hipFloatComplex*  d_out_tile = nullptr;            // output tiled
    hipFloatComplex*  d_fft_in   = nullptr;            // cuFFT: C2C input
    hipFloatComplex*  d_fft_out  = nullptr;            // cuFFT output
    CUDA_CHECK(hipMalloc(&d_img     , N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_out_tile, N * sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMalloc(&d_fft_in  , N * sizeof(hipFloatComplex)));
    CUDA_CHECK(hipMalloc(&d_fft_out , N * sizeof(hipFloatComplex)));

    // Host complex buffer for cuFFT
    std::vector<hipFloatComplex> h_img_c(N);
    for (size_t i = 0; i < N; ++i)
        h_img_c[i] = make_hipFloatComplex(h_img[i], 0.f);

    // CUDA events
    hipEvent_t H2D_tile_beg, H2D_tile_end,
                Kern_tile_beg, Kern_tile_end,
                D2H_tile_beg, D2H_tile_end,
                H2D_fft_beg,  H2D_fft_end,
                Kern_fft_beg, Kern_fft_end,
                D2H_fft_beg,  D2H_fft_end;
    hipEventCreate(&H2D_tile_beg);  hipEventCreate(&H2D_tile_end);
    hipEventCreate(&Kern_tile_beg); hipEventCreate(&Kern_tile_end);
    hipEventCreate(&D2H_tile_beg);  hipEventCreate(&D2H_tile_end);
    hipEventCreate(&H2D_fft_beg);   hipEventCreate(&H2D_fft_end);
    hipEventCreate(&Kern_fft_beg);  hipEventCreate(&Kern_fft_end);
    hipEventCreate(&D2H_fft_beg);   hipEventCreate(&D2H_fft_end);

    // Tiled path
    hipEventRecord(H2D_tile_beg);
    CUDA_CHECK(hipMemcpy(d_img, h_img.data(),
                          N * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(H2D_tile_end);

    dim3 block(16, 16);
    dim3 grid((W + 15) / 16, (H + 15) / 16);
    hipEventRecord(Kern_tile_beg);
    dft2d_kernel_tiled<<<grid, block>>>(d_img, d_out_tile, W, H);
    hipEventRecord(Kern_tile_end);

    std::vector<hipFloatComplex> h_out_tile(N);
    hipEventRecord(D2H_tile_beg);
    CUDA_CHECK(hipMemcpy(h_out_tile.data(), d_out_tile,
                          N * sizeof(hipFloatComplex),
                          hipMemcpyDeviceToHost));
    hipEventRecord(D2H_tile_end);

    // cuFFT path
    hipEventRecord(H2D_fft_beg);
    CUDA_CHECK(hipMemcpy(d_fft_in, h_img_c.data(),
                          N * sizeof(hipFloatComplex), hipMemcpyHostToDevice));
    hipEventRecord(H2D_fft_end);

    hipfftHandle plan;
    CUFFT_CHECK(hipfftPlan2d(&plan, H, W, HIPFFT_C2C));

    hipEventRecord(Kern_fft_beg);
    CUFFT_CHECK(hipfftExecC2C(plan, d_fft_in, d_fft_out, HIPFFT_FORWARD));
    hipEventRecord(Kern_fft_end);

    std::vector<hipFloatComplex> h_out_fft(N);
    hipEventRecord(D2H_fft_beg);
    CUDA_CHECK(hipMemcpy(h_out_fft.data(), d_fft_out,
                          N * sizeof(hipFloatComplex),
                          hipMemcpyDeviceToHost));
    hipEventRecord(D2H_fft_end);
    CUDA_CHECK(hipDeviceSynchronize());
    hipfftDestroy(plan);

    // Timing results
    float h2d_tile_ms, k_tile_ms, d2h_tile_ms;
    float h2d_fft_ms , k_fft_ms , d2h_fft_ms ;
    hipEventElapsedTime(&h2d_tile_ms, H2D_tile_beg, H2D_tile_end);
    hipEventElapsedTime(&k_tile_ms  , Kern_tile_beg, Kern_tile_end);
    hipEventElapsedTime(&d2h_tile_ms, D2H_tile_beg, D2H_tile_end);
    hipEventElapsedTime(&h2d_fft_ms , H2D_fft_beg , H2D_fft_end );
    hipEventElapsedTime(&k_fft_ms   , Kern_fft_beg, Kern_fft_end);
    hipEventElapsedTime(&d2h_fft_ms , D2H_fft_beg , D2H_fft_end);

    std::cout << std::fixed << std::setprecision(3);

    std::cout << "Tiled kernel:\n";
    std::cout << "GPU H2D     : " << h2d_tile_ms << " ms\n";
    std::cout << "GPU Kernel  : " << k_tile_ms   << " ms\n";
    std::cout << "GPU D2H     : " << d2h_tile_ms << " ms\n";
    std::cout << "GPU Total   : " << (h2d_tile_ms + k_tile_ms + d2h_tile_ms)
              << " ms\n\n";

    std::cout << "cuFFT       :\n";
    std::cout << "GPU H2D     : " << h2d_fft_ms  << " ms\n";
    std::cout << "GPU Kernel  : " << k_fft_ms    << " ms\n";
    std::cout << "GPU D2H     : " << d2h_fft_ms  << " ms\n";
    std::cout << "GPU Total   : " << (h2d_fft_ms + k_fft_ms + d2h_fft_ms)
              << " ms\n";

    // Clean-up
    hipFree(d_img); hipFree(d_out_tile);
    hipFree(d_fft_in); hipFree(d_fft_out);

    for (auto ev : {H2D_tile_beg, H2D_tile_end, Kern_tile_beg, Kern_tile_end,
                    D2H_tile_beg, D2H_tile_end, H2D_fft_beg,  H2D_fft_end,
                    Kern_fft_beg, Kern_fft_end, D2H_fft_beg,  D2H_fft_end})
        hipEventDestroy(ev);

    return 0;
}

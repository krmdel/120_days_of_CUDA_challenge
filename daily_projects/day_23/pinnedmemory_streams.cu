#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define cudaCheckError(call) {                                             \
    hipError_t err = call;                                                \
    if( err != hipSuccess ) {                                             \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__,  \
                hipGetErrorString(err));                                  \
        exit(EXIT_FAILURE);                                                \
    }                                                                      \
}

__global__ void vectorAdd(const float *A, const float *B, float *C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
        C[i] = A[i] + B[i];
}

int main(void) {
    const int totalElements = 1 << 20;
    const int bytes = totalElements * sizeof(float);
    const int nStreams = 4;
    const int chunkSize = totalElements / nStreams;
    const int chunkBytes = chunkSize * sizeof(float);
    
    // Allocate pinned host memory for input and output arrays (streaming)
    float *h_A, *h_B, *h_C;
    cudaCheckError(hipHostMalloc((void**)&h_A, bytes, hipHostMallocDefault));
    cudaCheckError(hipHostMalloc((void**)&h_B, bytes, hipHostMallocDefault));
    cudaCheckError(hipHostMalloc((void**)&h_C, bytes, hipHostMallocDefault));
    
    // Initialize input data on the host (streaming)
    for (int i = 0; i < totalElements; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }
    
    // Allocate device memory for vectors (streaming)
    float *d_A, *d_B, *d_C;
    cudaCheckError(hipMalloc((void**)&d_A, bytes));
    cudaCheckError(hipMalloc((void**)&d_B, bytes));
    cudaCheckError(hipMalloc((void**)&d_C, bytes));
    
    // Create CUDA streams and per-stream events for timing (streaming)
    hipStream_t streams[nStreams];
    hipEvent_t startEvents[nStreams], htodEvents[nStreams], kernelEvents[nStreams], dtohEvents[nStreams];
    for (int i = 0; i < nStreams; i++) {
        cudaCheckError(hipStreamCreate(&streams[i]));
        cudaCheckError(hipEventCreate(&startEvents[i]));
        cudaCheckError(hipEventCreate(&htodEvents[i]));
        cudaCheckError(hipEventCreate(&kernelEvents[i]));
        cudaCheckError(hipEventCreate(&dtohEvents[i]));
    }
    
    hipEvent_t overallStart, overallStop;
    cudaCheckError(hipEventCreate(&overallStart));
    cudaCheckError(hipEventCreate(&overallStop));
    
    cudaCheckError(hipEventRecord(overallStart, 0));
    
    const int threadsPerBlock = 256;
    
    // Enqueue asynchronous operations in each stream (streaming)
    for (int i = 0; i < nStreams; i++) {
        int offset = i * chunkSize;
        cudaCheckError(hipEventRecord(startEvents[i], streams[i]));
        
        cudaCheckError(hipMemcpyAsync(d_A + offset, h_A + offset, chunkBytes, hipMemcpyHostToDevice, streams[i]));
        cudaCheckError(hipMemcpyAsync(d_B + offset, h_B + offset, chunkBytes, hipMemcpyHostToDevice, streams[i]));
        cudaCheckError(hipEventRecord(htodEvents[i], streams[i]));
        
        int blocks = (chunkSize + threadsPerBlock - 1) / threadsPerBlock;
        vectorAdd<<<blocks, threadsPerBlock, 0, streams[i]>>>(d_A + offset, d_B + offset, d_C + offset, chunkSize);
        cudaCheckError(hipEventRecord(kernelEvents[i], streams[i]));
        
        cudaCheckError(hipMemcpyAsync(h_C + offset, d_C + offset, chunkBytes, hipMemcpyDeviceToHost, streams[i]));
        cudaCheckError(hipEventRecord(dtohEvents[i], streams[i]));
    }
    
    cudaCheckError(hipDeviceSynchronize());
    
    cudaCheckError(hipEventRecord(overallStop, 0));
    cudaCheckError(hipEventSynchronize(overallStop));
    
    float maxHtod = 0, maxKernel = 0, maxDtoh = 0, maxTotal = 0;
    for (int i = 0; i < nStreams; i++) {
        float t_htod = 0, t_kernel = 0, t_dtoh = 0, t_total = 0;
        cudaCheckError(hipEventElapsedTime(&t_htod, startEvents[i], htodEvents[i]));
        cudaCheckError(hipEventElapsedTime(&t_kernel, htodEvents[i], kernelEvents[i]));
        cudaCheckError(hipEventElapsedTime(&t_dtoh, kernelEvents[i], dtohEvents[i]));
        cudaCheckError(hipEventElapsedTime(&t_total, startEvents[i], dtohEvents[i]));
        
        if(t_htod > maxHtod) maxHtod = t_htod;
        if(t_kernel > maxKernel) maxKernel = t_kernel;
        if(t_dtoh > maxDtoh) maxDtoh = t_dtoh;
        if(t_total > maxTotal) maxTotal = t_total;
    }
    
    float overallTime = 0;
    cudaCheckError(hipEventElapsedTime(&overallTime, overallStart, overallStop));
    
    printf("Streaming (pinned memory) timings:\n");
    printf("Max host to device copy time per stream: %f ms\n", maxHtod);
    printf("Max kernel execution time per stream: %f ms\n", maxKernel);
    printf("Max device to host copy time per stream: %f ms\n", maxDtoh);
    printf("Max total time per stream: %f ms\n", maxTotal);
    printf("Overall elapsed time: %f ms\n", overallTime);
       
    // Baseline
    float *h_A_sync = (float*)malloc(bytes);
    float *h_B_sync = (float*)malloc(bytes);
    float *h_C_sync = (float*)malloc(bytes);
    for (int i = 0; i < totalElements; i++) {
        h_A_sync[i] = static_cast<float>(i);
        h_B_sync[i] = static_cast<float>(i * 2);
    }
    
    float *d_A_sync, *d_B_sync, *d_C_sync;
    cudaCheckError(hipMalloc((void**)&d_A_sync, bytes));
    cudaCheckError(hipMalloc((void**)&d_B_sync, bytes));
    cudaCheckError(hipMalloc((void**)&d_C_sync, bytes));
    
    hipEvent_t baseStart, baseHtoD, baseKernel, baseDtoH, baseStop;
    cudaCheckError(hipEventCreate(&baseStart));
    cudaCheckError(hipEventCreate(&baseHtoD));
    cudaCheckError(hipEventCreate(&baseKernel));
    cudaCheckError(hipEventCreate(&baseDtoH));
    cudaCheckError(hipEventCreate(&baseStop));
    
    cudaCheckError(hipEventRecord(baseStart, 0));
    cudaCheckError(hipMemcpy(d_A_sync, h_A_sync, bytes, hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(d_B_sync, h_B_sync, bytes, hipMemcpyHostToDevice));
    cudaCheckError(hipEventRecord(baseHtoD, 0));
    
    int blocks = (totalElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocks, threadsPerBlock>>>(d_A_sync, d_B_sync, d_C_sync, totalElements);
    cudaCheckError(hipEventRecord(baseKernel, 0));
    
    cudaCheckError(hipMemcpy(h_C_sync, d_C_sync, bytes, hipMemcpyDeviceToHost));
    cudaCheckError(hipEventRecord(baseDtoH, 0));
    
    cudaCheckError(hipEventRecord(baseStop, 0));
    cudaCheckError(hipEventSynchronize(baseStop));
    
    float baseHtoDTime = 0, baseKernelTime = 0, baseDtoHTime = 0, baseTotalTime = 0;
    cudaCheckError(hipEventElapsedTime(&baseHtoDTime, baseStart, baseHtoD));
    cudaCheckError(hipEventElapsedTime(&baseKernelTime, baseHtoD, baseKernel));
    cudaCheckError(hipEventElapsedTime(&baseDtoHTime, baseKernel, baseDtoH));
    cudaCheckError(hipEventElapsedTime(&baseTotalTime, baseStart, baseStop));
    
    printf("Baseline timings:\n");
    printf("Host to device copy time: %f ms\n", baseHtoDTime);
    printf("Kernel execution time: %f ms\n", baseKernelTime);
    printf("Device to host copy time: %f ms\n", baseDtoHTime);
    printf("Total time: %f ms\n", baseTotalTime);
    
    // Clean up streaming
    for (int i = 0; i < nStreams; i++) {
        cudaCheckError(hipEventDestroy(startEvents[i]));
        cudaCheckError(hipEventDestroy(htodEvents[i]));
        cudaCheckError(hipEventDestroy(kernelEvents[i]));
        cudaCheckError(hipEventDestroy(dtohEvents[i]));
        cudaCheckError(hipStreamDestroy(streams[i]));
    }
    cudaCheckError(hipEventDestroy(overallStart));
    cudaCheckError(hipEventDestroy(overallStop));
    
    cudaCheckError(hipFree(d_A));
    cudaCheckError(hipFree(d_B));
    cudaCheckError(hipFree(d_C));
    cudaCheckError(hipHostFree(h_A));
    cudaCheckError(hipHostFree(h_B));
    cudaCheckError(hipHostFree(h_C));
    
    // Clean up baseline
    cudaCheckError(hipEventDestroy(baseStart));
    cudaCheckError(hipEventDestroy(baseHtoD));
    cudaCheckError(hipEventDestroy(baseKernel));
    cudaCheckError(hipEventDestroy(baseDtoH));
    cudaCheckError(hipEventDestroy(baseStop));
    
    cudaCheckError(hipFree(d_A_sync));
    cudaCheckError(hipFree(d_B_sync));
    cudaCheckError(hipFree(d_C_sync));
    free(h_A_sync);
    free(h_B_sync);
    free(h_C_sync);
    
    return 0;
}

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <float.h>  // for FLT_MAX

// Image Parameters
#define IMAGE_WIDTH    32
#define IMAGE_HEIGHT   32
#define IMAGE_CHANNELS 3

// CNN input parameters
#define IN_WIDTH     IMAGE_WIDTH
#define IN_HEIGHT    IMAGE_HEIGHT
#define IN_CHANNELS  IMAGE_CHANNELS

// Network parameters
#define OUT_CHANNELS 1
#define KERNEL_SIZE  3
#define STRIDE       2

#define OUT_HEIGHT ((IN_HEIGHT - KERNEL_SIZE) / STRIDE + 1)
#define OUT_WIDTH  ((IN_WIDTH  - KERNEL_SIZE) / STRIDE + 1)

// Pooling parameters
#define POOL_SIZE   2
#define POOL_STRIDE 2
#define POOL_OUT_HEIGHT ((OUT_HEIGHT - POOL_SIZE) / POOL_STRIDE + 1)
#define POOL_OUT_WIDTH  ((OUT_WIDTH  - POOL_SIZE) / POOL_STRIDE + 1)

// GPU Kernels:

// Convolution kernel
__global__ void conv_forward(const float* input, const float* weight, float* output,
                             int in_channels, int out_channels,
                             int in_h, int in_w, int kernel_h, int kernel_w, int stride) {
    int ox = blockIdx.x * blockDim.x + threadIdx.x;
    int oy = blockIdx.y * blockDim.y + threadIdx.y;
    int oc = blockIdx.z;
    if (ox < OUT_WIDTH && oy < OUT_HEIGHT) {
        float sum = 0.0f;
        for (int ic = 0; ic < in_channels; ic++) {
            for (int i = 0; i < kernel_h; i++) {
                for (int j = 0; j < kernel_w; j++) {
                    int in_y = oy * stride + i;
                    int in_x = ox * stride + j;
                    int input_idx = ic * in_h * in_w + in_y * in_w + in_x;
                    int weight_idx = oc * in_channels * kernel_h * kernel_w +
                                     ic * kernel_h * kernel_w + i * kernel_w + j;
                    sum += input[input_idx] * weight[weight_idx];
                }
            }
        }
        int output_idx = oc * OUT_HEIGHT * OUT_WIDTH + oy * OUT_WIDTH + ox;
        output[output_idx] = sum;
    }
}

// Kernel to compute gradients for the convolution weights
__global__ void conv_weight_grad(const float* input, const float* d_output, float* d_weight,
                                   int in_channels, int out_channels,
                                   int in_h, int in_w, int kernel_h, int kernel_w, int stride) {
    int oc = blockIdx.x; // output channel index
    int ic = blockIdx.y; // input channel index
    int i = threadIdx.y; // kernel row index
    int j = threadIdx.x; // kernel column index
    if (i < kernel_h && j < kernel_w) {
        float grad = 0.0f;
        for (int oy = 0; oy < OUT_HEIGHT; oy++) {
            for (int ox = 0; ox < OUT_WIDTH; ox++) {
                int in_y = oy * stride + i;
                int in_x = ox * stride + j;
                int input_idx = ic * in_h * in_w + in_y * in_w + in_x;
                int output_idx = oc * OUT_HEIGHT * OUT_WIDTH + oy * OUT_WIDTH + ox;
                grad += input[input_idx] * d_output[output_idx];
            }
        }
        int weight_idx = oc * in_channels * kernel_h * kernel_w +
                         ic * kernel_h * kernel_w + i * kernel_w + j;
        d_weight[weight_idx] = grad;
    }
}

// Kernel to update weights on GPU using gradient descent
__global__ void update_weights(float* weight, const float* d_weight, float learning_rate, int weight_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < weight_size) {
        weight[idx] = weight[idx] - learning_rate * d_weight[idx];
    }
}

// Max Pooling Forward Kernel
__global__ void max_pool_forward(const float* input, float* output, 
                                 int channels, int in_h, int in_w, 
                                 int pool_size, int pool_stride) {
    int ow = blockIdx.x * blockDim.x + threadIdx.x;
    int oh = blockIdx.y * blockDim.y + threadIdx.y;
    int c  = blockIdx.z;
    if (ow < POOL_OUT_WIDTH && oh < POOL_OUT_HEIGHT) {
        float max_val = -FLT_MAX;
        for (int i = 0; i < pool_size; i++) {
            for (int j = 0; j < pool_size; j++) {
                int in_y = oh * pool_stride + i;
                int in_x = ow * pool_stride + j;
                int index = c * in_h * in_w + in_y * in_w + in_x;
                if (input[index] > max_val)
                    max_val = input[index];
            }
        }
        int out_index = c * POOL_OUT_HEIGHT * POOL_OUT_WIDTH + oh * POOL_OUT_WIDTH + ow;
        output[out_index] = max_val;
    }
}

// Average Pooling Forward Kernel
__global__ void avg_pool_forward(const float* input, float* output, 
                                 int channels, int in_h, int in_w, 
                                 int pool_size, int pool_stride) {
    int ow = blockIdx.x * blockDim.x + threadIdx.x;
    int oh = blockIdx.y * blockDim.y + threadIdx.y;
    int c  = blockIdx.z;
    if (ow < POOL_OUT_WIDTH && oh < POOL_OUT_HEIGHT) {
        float sum = 0.0f;
        for (int i = 0; i < pool_size; i++) {
            for (int j = 0; j < pool_size; j++) {
                int in_y = oh * pool_stride + i;
                int in_x = ow * pool_stride + j;
                int index = c * in_h * in_w + in_y * in_w + in_x;
                sum += input[index];
            }
        }
        int out_index = c * POOL_OUT_HEIGHT * POOL_OUT_WIDTH + oh * POOL_OUT_WIDTH + ow;
        output[out_index] = sum / (pool_size * pool_size);
    }
}

// Batch Normalization Forward Kernel
__global__ void batch_norm_forward(const float* input, float* output, 
                                   int channels, int size, 
                                   const float* gamma, const float* beta, float epsilon) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if (c < channels) {
        float sum = 0.0f;
        for (int i = 0; i < size; i++){
            int index = c * size + i;
            sum += input[index];
        }
        float mean = sum / size;
        float var = 0.0f;
        for (int i = 0; i < size; i++){
            int index = c * size + i;
            float diff = input[index] - mean;
            var += diff * diff;
        }
        var /= size;
        float inv_std = 1.0f / sqrtf(var + epsilon);
        for (int i = 0; i < size; i++){
            int index = c * size + i;
            output[index] = gamma[c] * (input[index] - mean) * inv_std + beta[c];
        }
    }
}

// Dropout Forward Kernel
__global__ void dropout_forward(const float* input, float* output, int size, float dropout_prob, unsigned int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        unsigned int x = seed ^ idx;
        x = (x * 1664525u + 1013904223u);
        float rand_val = (float)(x & 0x00FFFFFF) / (float)0x01000000;
        if (rand_val < dropout_prob)
            output[idx] = 0.0f;
        else
            output[idx] = input[idx] / (1.0f - dropout_prob);
    }
}

// CPU Functions:

// Convolution
void conv_forward_cpu(const float* input, const float* weight, float* output,
                      int in_channels, int out_channels,
                      int in_h, int in_w, int kernel_h, int kernel_w, int stride) {
    for (int oc = 0; oc < out_channels; oc++) {
        for (int oy = 0; oy < OUT_HEIGHT; oy++) {
            for (int ox = 0; ox < OUT_WIDTH; ox++) {
                float sum = 0.0f;
                for (int ic = 0; ic < in_channels; ic++) {
                    for (int i = 0; i < kernel_h; i++) {
                        for (int j = 0; j < kernel_w; j++) {
                            int in_y = oy * stride + i;
                            int in_x = ox * stride + j;
                            int input_idx = ic * in_h * in_w + in_y * in_w + in_x;
                            int weight_idx = oc * in_channels * kernel_h * kernel_w +
                                             ic * kernel_h * kernel_w + i * kernel_w + j;
                            sum += input[input_idx] * weight[weight_idx];
                        }
                    }
                }
                int output_idx = oc * OUT_HEIGHT * OUT_WIDTH + oy * OUT_WIDTH + ox;
                output[output_idx] = sum;
            }
        }
    }
}

// Gradient computation for convolution weights
void conv_weight_grad_cpu(const float* input, const float* d_output, float* d_weight,
                          int in_channels, int out_channels,
                          int in_h, int in_w, int kernel_h, int kernel_w, int stride) {
    for (int oc = 0; oc < out_channels; oc++) {
        for (int ic = 0; ic < in_channels; ic++) {
            for (int i = 0; i < kernel_h; i++) {
                for (int j = 0; j < kernel_w; j++) {
                    float grad = 0.0f;
                    for (int oy = 0; oy < OUT_HEIGHT; oy++) {
                        for (int ox = 0; ox < OUT_WIDTH; ox++){
                            int in_y = oy * stride + i;
                            int in_x = ox * stride + j;
                            int input_idx = ic * in_h * in_w + in_y * in_w + in_x;
                            int output_idx = oc * OUT_HEIGHT * OUT_WIDTH + oy * OUT_WIDTH + ox;
                            grad += input[input_idx] * d_output[output_idx];
                        }
                    }
                    int weight_idx = oc * in_channels * kernel_h * kernel_w +
                                     ic * kernel_h * kernel_w + i * kernel_w + j;
                    d_weight[weight_idx] = grad;
                }
            }
        }
    }
}

// Weight update
void update_weights_cpu(float* weight, const float* d_weight, float learning_rate, int weight_size) {
    for (int idx = 0; idx < weight_size; idx++) {
        weight[idx] = weight[idx] - learning_rate * d_weight[idx];
    }
}

// Max Pooling Forward
void max_pool_forward_cpu(const float* input, float* output, int channels, int in_h, int in_w, int pool_size, int pool_stride) {
    for (int c = 0; c < channels; c++){
        for (int oh = 0; oh < POOL_OUT_HEIGHT; oh++){
            for (int ow = 0; ow < POOL_OUT_WIDTH; ow++){
                float max_val = -FLT_MAX;
                for (int i = 0; i < pool_size; i++){
                    for (int j = 0; j < pool_size; j++){
                        int in_y = oh * pool_stride + i;
                        int in_x = ow * pool_stride + j;
                        int index = c * in_h * in_w + in_y * in_w + in_x;
                        if (input[index] > max_val)
                            max_val = input[index];
                    }
                }
                int out_index = c * POOL_OUT_HEIGHT * POOL_OUT_WIDTH + oh * POOL_OUT_WIDTH + ow;
                output[out_index] = max_val;
            }
        }
    }
}

// Average Pooling Forward
void avg_pool_forward_cpu(const float* input, float* output, int channels, int in_h, int in_w, int pool_size, int pool_stride) {
    for (int c = 0; c < channels; c++){
        for (int oh = 0; oh < POOL_OUT_HEIGHT; oh++){
            for (int ow = 0; ow < POOL_OUT_WIDTH; ow++){
                float sum = 0.0f;
                for (int i = 0; i < pool_size; i++){
                    for (int j = 0; j < pool_size; j++){
                        int in_y = oh * pool_stride + i;
                        int in_x = ow * pool_stride + j;
                        int index = c * in_h * in_w + in_y * in_w + in_x;
                        sum += input[index];
                    }
                }
                int out_index = c * POOL_OUT_HEIGHT * POOL_OUT_WIDTH + oh * POOL_OUT_WIDTH + ow;
                output[out_index] = sum / (pool_size * pool_size);
            }
        }
    }
}

// Batch Normalization Forward
void batch_norm_forward_cpu(const float* input, float* output, int channels, int size, const float* gamma, const float* beta, float epsilon) {
    for (int c = 0; c < channels; c++) {
       float sum = 0.0f;
       for (int i = 0; i < size; i++){
           int index = c * size + i;
           sum += input[index];
       }
       float mean = sum / size;
       float var = 0.0f;
       for (int i = 0; i < size; i++){
           int index = c * size + i;
           float diff = input[index] - mean;
           var += diff * diff;
       }
       var /= size;
       float inv_std = 1.0f / sqrtf(var + epsilon);
       for (int i = 0; i < size; i++){
           int index = c * size + i;
           output[index] = gamma[c] * (input[index] - mean) * inv_std + beta[c];
       }
    }
}

// Dropout Forward
void dropout_forward_cpu(const float* input, float* output, int size, float dropout_prob) {
    for (int i = 0; i < size; i++) {
        float rand_val = (float)rand() / RAND_MAX;
        if (rand_val < dropout_prob)
            output[i] = 0.0f;
        else
            output[i] = input[i] / (1.0f - dropout_prob);
    }
}

int main(){
    srand(time(NULL));

    int input_size  = IN_CHANNELS * IN_HEIGHT * IN_WIDTH;
    int weight_size = OUT_CHANNELS * IN_CHANNELS * KERNEL_SIZE * KERNEL_SIZE;
    int conv_output_size = OUT_CHANNELS * OUT_HEIGHT * OUT_WIDTH;
    int pool_output_size = OUT_CHANNELS * POOL_OUT_HEIGHT * POOL_OUT_WIDTH;

    // Allocate host memory for convolution
    float* h_input    = (float*)malloc(input_size * sizeof(float));
    float* h_weight   = (float*)malloc(weight_size * sizeof(float));
    float* h_output   = (float*)malloc(conv_output_size * sizeof(float));

    // Allocate host memory for gradients
    float* h_d_output = (float*)malloc(conv_output_size * sizeof(float));

    // Allocate host memory for new layers (pooling, batch norm, dropout)
    float* h_pool_output   = (float*)malloc(pool_output_size * sizeof(float));
    float* h_bn_output     = (float*)malloc(pool_output_size * sizeof(float));
    float* h_dropout_output= (float*)malloc(pool_output_size * sizeof(float));

    // Initialize input and weights
    for (int i = 0; i < input_size; i++){
        h_input[i] = (float)rand() / RAND_MAX;
    }
    for (int i = 0; i < weight_size; i++){
        h_weight[i] = ((float)rand() / RAND_MAX) - 0.5f;
    }

    // Parameters for batch normalization and dropout
    float dropout_prob = 0.5f;
    float epsilon = 1e-5f;
    float* h_gamma = (float*)malloc(OUT_CHANNELS * sizeof(float));
    float* h_beta  = (float*)malloc(OUT_CHANNELS * sizeof(float));
    for (int i = 0; i < OUT_CHANNELS; i++){
        h_gamma[i] = 1.0f;
        h_beta[i]  = 0.0f;
    }

    // Make a separate copy of initial weights for CPU training
    float* cpu_weight = (float*)malloc(weight_size * sizeof(float));
    memcpy(cpu_weight, h_weight, weight_size * sizeof(float));

    int num_epochs = 10;
    float learning_rate = 0.01f;
    
    printf("Starting CPU Training for CNN (convolution only)...\n");
    clock_t cpu_train_start = clock();
    double total_cpu_epoch_time = 0.0;
    for (int epoch = 0; epoch < num_epochs; epoch++){
        clock_t epoch_start = clock();
        // CPU forward pass
        conv_forward_cpu(h_input, cpu_weight, h_output,
                         IN_CHANNELS, OUT_CHANNELS,
                         IN_HEIGHT, IN_WIDTH, KERNEL_SIZE, KERNEL_SIZE, STRIDE);
        // Compute loss and output gradient (loss computed on conv output)
        float loss = 0.0f;
        for (int i = 0; i < conv_output_size; i++){
            float diff = h_output[i];
            loss += diff * diff;
            h_d_output[i] = 2.0f * diff / conv_output_size;
        }
        loss /= conv_output_size;
        // Compute weight gradient on CPU (for convolution)
        float* cpu_d_weight = (float*)malloc(weight_size * sizeof(float));
        conv_weight_grad_cpu(h_input, h_d_output, cpu_d_weight,
                             IN_CHANNELS, OUT_CHANNELS,
                             IN_HEIGHT, IN_WIDTH, KERNEL_SIZE, KERNEL_SIZE, STRIDE);
        // Update weights on CPU
        update_weights_cpu(cpu_weight, cpu_d_weight, learning_rate, weight_size);
        free(cpu_d_weight);
        clock_t epoch_end = clock();
        double epoch_time = ((double)(epoch_end - epoch_start)) / CLOCKS_PER_SEC * 1000.0;
        total_cpu_epoch_time += epoch_time;
        printf("CPU Epoch %d/%d Loss: %f Epoch Time (CPU): %.4f ms\n", epoch+1, num_epochs, loss, epoch_time);
    }
    clock_t cpu_train_end = clock();
    double total_cpu_train_time = ((double)(cpu_train_end - cpu_train_start)) / CLOCKS_PER_SEC * 1000.0;

    // CPU Inference including new layers (pooling, batch norm, dropout)
    clock_t cpu_inf_start = clock();
    // Convolution forward
    conv_forward_cpu(h_input, cpu_weight, h_output,
                     IN_CHANNELS, OUT_CHANNELS,
                     IN_HEIGHT, IN_WIDTH, KERNEL_SIZE, KERNEL_SIZE, STRIDE);
    // Pooling
    max_pool_forward_cpu(h_output, h_pool_output, OUT_CHANNELS, OUT_HEIGHT, OUT_WIDTH, POOL_SIZE, POOL_STRIDE);
    // Batch Normalization
    batch_norm_forward_cpu(h_pool_output, h_bn_output, OUT_CHANNELS, POOL_OUT_HEIGHT*POOL_OUT_WIDTH, h_gamma, h_beta, epsilon);
    // Dropout
    dropout_forward_cpu(h_bn_output, h_dropout_output, pool_output_size, dropout_prob);
    clock_t cpu_inf_end = clock();
    double cpu_inf_time = ((double)(cpu_inf_end - cpu_inf_start)) / CLOCKS_PER_SEC * 1000.0;

    // GPU Training and inference for convolution (unchanged from your original code)
    float *d_input, *d_weight, *d_output, *d_d_output, *d_d_weight;
    hipMalloc((void**)&d_input,    input_size * sizeof(float));
    hipMalloc((void**)&d_weight,   weight_size * sizeof(float));
    hipMalloc((void**)&d_output,   conv_output_size * sizeof(float));
    hipMalloc((void**)&d_d_output, conv_output_size * sizeof(float));
    hipMalloc((void**)&d_d_weight, weight_size * sizeof(float));

    // Copy input and weights to GPU and measure Host->Device copy time
    hipEvent_t start_copy, stop_copy;
    hipEventCreate(&start_copy);
    hipEventCreate(&stop_copy);
    hipEventRecord(start_copy, 0);
    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_weight, h_weight, weight_size * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    hipEventRecord(stop_copy, 0);
    hipEventSynchronize(stop_copy);
    float copy_h2d_time;
    hipEventElapsedTime(&copy_h2d_time, start_copy, stop_copy);
    hipEventDestroy(start_copy);
    hipEventDestroy(stop_copy);

    // GPU kernel parameters for convolution
    dim3 blockSize(16, 16, 1);
    dim3 gridSize((OUT_WIDTH + blockSize.x - 1) / blockSize.x,
                  (OUT_HEIGHT + blockSize.y - 1) / blockSize.y,
                  OUT_CHANNELS);
    dim3 gridSizeGrad(OUT_CHANNELS, IN_CHANNELS, 1);
    dim3 blockSizeGrad(KERNEL_SIZE, KERNEL_SIZE, 1);
    int threadsPerBlock = 256;
    int blocksPerGrid = (weight_size + threadsPerBlock - 1) / threadsPerBlock;

    // GPU training:
    float total_gpu_kernel_time = 0.0f;
    float total_gpu_h2d_time = 0.0f;
    float total_gpu_d2h_time = 0.0f;

    hipEvent_t gpu_train_start, gpu_train_end;
    hipEventCreate(&gpu_train_start);
    hipEventCreate(&gpu_train_end);
    hipEventRecord(gpu_train_start, 0);

    printf("\nStarting GPU Training for CNN (convolution only)...\n");
    for (int epoch = 0; epoch < num_epochs; epoch++){
        clock_t gpu_epoch_start = clock();

        // Forward pass kernel
        hipEvent_t start_k, end_k;
        hipEventCreate(&start_k);
        hipEventCreate(&end_k);
        hipEventRecord(start_k, 0);
        conv_forward<<<gridSize, blockSize>>>(d_input, d_weight, d_output,
                                              IN_CHANNELS, OUT_CHANNELS,
                                              IN_HEIGHT, IN_WIDTH, KERNEL_SIZE, KERNEL_SIZE, STRIDE);
        hipDeviceSynchronize();
        hipEventRecord(end_k, 0);
        hipEventSynchronize(end_k);
        float t_conv = 0.0f;
        hipEventElapsedTime(&t_conv, start_k, end_k);
        total_gpu_kernel_time += t_conv;
        hipEventDestroy(start_k);
        hipEventDestroy(end_k);

        // Copy convolution output from GPU (Device -> Host) for loss computation
        hipEvent_t start_d2h, end_d2h;
        hipEventCreate(&start_d2h);
        hipEventCreate(&end_d2h);
        hipEventRecord(start_d2h, 0);
        hipMemcpy(h_output, d_output, conv_output_size * sizeof(float), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipEventRecord(end_d2h, 0);
        hipEventSynchronize(end_d2h);
        float t_d2h = 0.0f;
        hipEventElapsedTime(&t_d2h, start_d2h, end_d2h);
        total_gpu_d2h_time += t_d2h;
        hipEventDestroy(start_d2h);
        hipEventDestroy(end_d2h);

        // Loss and gradient computation on CPU (for convolution output)
        float loss = 0.0f;
        for (int i = 0; i < conv_output_size; i++){
            float diff = h_output[i];
            loss += diff * diff;
            h_d_output[i] = 2.0f * diff / conv_output_size;
        }
        loss /= conv_output_size;

        // Copy gradient to GPU (Host -> Device)
        hipEvent_t start_h2d, end_h2d;
        hipEventCreate(&start_h2d);
        hipEventCreate(&end_h2d);
        hipEventRecord(start_h2d, 0);
        hipMemcpy(d_d_output, h_d_output, conv_output_size * sizeof(float), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        hipEventRecord(end_h2d, 0);
        hipEventSynchronize(end_h2d);
        float t_h2d = 0.0f;
        hipEventElapsedTime(&t_h2d, start_h2d, end_h2d);
        total_gpu_h2d_time += t_h2d;
        hipEventDestroy(start_h2d);
        hipEventDestroy(end_h2d);

        // Backward pass: compute weight gradient
        hipEventCreate(&start_k);
        hipEventCreate(&end_k);
        hipEventRecord(start_k, 0);
        conv_weight_grad<<<gridSizeGrad, blockSizeGrad>>>(d_input, d_d_output, d_d_weight,
                                                          IN_CHANNELS, OUT_CHANNELS,
                                                          IN_HEIGHT, IN_WIDTH, KERNEL_SIZE, KERNEL_SIZE, STRIDE);
        hipDeviceSynchronize();
        hipEventRecord(end_k, 0);
        hipEventSynchronize(end_k);
        float t_grad = 0.0f;
        hipEventElapsedTime(&t_grad, start_k, end_k);
        total_gpu_kernel_time += t_grad;
        hipEventDestroy(start_k);
        hipEventDestroy(end_k);

        // Update weights kernel
        hipEventCreate(&start_k);
        hipEventCreate(&end_k);
        hipEventRecord(start_k, 0);
        update_weights<<<blocksPerGrid, threadsPerBlock>>>(d_weight, d_d_weight, learning_rate, weight_size);
        hipDeviceSynchronize();
        hipEventRecord(end_k, 0);
        hipEventSynchronize(end_k);
        float t_update = 0.0f;
        hipEventElapsedTime(&t_update, start_k, end_k);
        total_gpu_kernel_time += t_update;
        hipEventDestroy(start_k);
        hipEventDestroy(end_k);

        clock_t gpu_epoch_end = clock();
        double epoch_cpu_time = ((double)(gpu_epoch_end - gpu_epoch_start)) / CLOCKS_PER_SEC * 1000.0;
        printf("GPU Epoch %d/%d Loss: %f Epoch Time (CPU): %.4f ms\n", epoch+1, num_epochs, loss, epoch_cpu_time);
    }
    hipEventRecord(gpu_train_end, 0);
    hipEventSynchronize(gpu_train_end);
    float total_gpu_train_time = 0.0f;
    hipEventElapsedTime(&total_gpu_train_time, gpu_train_start, gpu_train_end);
    hipEventDestroy(gpu_train_start);
    hipEventDestroy(gpu_train_end);

    // GPU Inference including new layers (Pooling, Batch Norm, Dropout)

    // Allocate device memory for new layers
    float *d_pool_output, *d_bn_output, *d_dropout_output;
    hipMalloc((void**)&d_pool_output, pool_output_size * sizeof(float));
    hipMalloc((void**)&d_bn_output, pool_output_size * sizeof(float));
    hipMalloc((void**)&d_dropout_output, pool_output_size * sizeof(float));

    // Allocate and copy batch norm parameters to device
    float *d_gamma, *d_beta;
    hipMalloc((void**)&d_gamma, OUT_CHANNELS * sizeof(float));
    hipMalloc((void**)&d_beta,  OUT_CHANNELS * sizeof(float));
    hipMemcpy(d_gamma, h_gamma, OUT_CHANNELS * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_beta, h_beta, OUT_CHANNELS * sizeof(float), hipMemcpyHostToDevice);

    unsigned int dropout_seed = time(NULL);

    // GPU Inference for convolution
    hipEvent_t inf_start_h2d, inf_end_h2d;
    hipEventCreate(&inf_start_h2d);
    hipEventCreate(&inf_end_h2d);
    hipEventRecord(inf_start_h2d, 0);
    hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    hipEventRecord(inf_end_h2d, 0);
    hipEventSynchronize(inf_end_h2d);
    float inf_h2d_time = 0.0f;
    hipEventElapsedTime(&inf_h2d_time, inf_start_h2d, inf_end_h2d);
    hipEventDestroy(inf_start_h2d);
    hipEventDestroy(inf_end_h2d);

    // Convolution inference kernel execution
    hipEvent_t inf_start_kernel, inf_end_kernel;
    hipEventCreate(&inf_start_kernel);
    hipEventCreate(&inf_end_kernel);
    hipEventRecord(inf_start_kernel, 0);
    conv_forward<<<gridSize, blockSize>>>(d_input, d_weight, d_output,
                                          IN_CHANNELS, OUT_CHANNELS,
                                          IN_HEIGHT, IN_WIDTH, KERNEL_SIZE, KERNEL_SIZE, STRIDE);
    hipDeviceSynchronize();
    hipEventRecord(inf_end_kernel, 0);
    hipEventSynchronize(inf_end_kernel);
    float inf_kernel_time = 0.0f;
    hipEventElapsedTime(&inf_kernel_time, inf_start_kernel, inf_end_kernel);
    hipEventDestroy(inf_start_kernel);
    hipEventDestroy(inf_end_kernel);

    // GPU: Pooling, Batch Norm and Dropout
    dim3 poolBlockSize(16, 16, 1);
    dim3 poolGridSize((POOL_OUT_WIDTH + poolBlockSize.x - 1) / poolBlockSize.x,
                      (POOL_OUT_HEIGHT + poolBlockSize.y - 1) / poolBlockSize.y,
                      OUT_CHANNELS);
    // Max Pooling kernel
    hipEvent_t pool_start, pool_end;
    hipEventCreate(&pool_start);
    hipEventCreate(&pool_end);
    hipEventRecord(pool_start, 0);
    max_pool_forward<<<poolGridSize, poolBlockSize>>>(d_output, d_pool_output,
                                                      OUT_CHANNELS, OUT_HEIGHT, OUT_WIDTH, POOL_SIZE, POOL_STRIDE);
    hipDeviceSynchronize();
    hipEventRecord(pool_end, 0);
    hipEventSynchronize(pool_end);
    float t_pool = 0.0f;
    hipEventElapsedTime(&t_pool, pool_start, pool_end);
    hipEventDestroy(pool_start);
    hipEventDestroy(pool_end);

    // Batch Norm kernel execution
    int bn_size = POOL_OUT_HEIGHT * POOL_OUT_WIDTH;
    int bnThreads = 32;
    int bnBlocks = (OUT_CHANNELS + bnThreads - 1) / bnThreads;
    hipEvent_t bn_start, bn_end;
    hipEventCreate(&bn_start);
    hipEventCreate(&bn_end);
    hipEventRecord(bn_start, 0);
    batch_norm_forward<<<bnBlocks, bnThreads>>>(d_pool_output, d_bn_output, OUT_CHANNELS, bn_size, d_gamma, d_beta, epsilon);
    hipDeviceSynchronize();
    hipEventRecord(bn_end, 0);
    hipEventSynchronize(bn_end);
    float t_bn = 0.0f;
    hipEventElapsedTime(&t_bn, bn_start, bn_end);
    hipEventDestroy(bn_start);
    hipEventDestroy(bn_end);

    // Dropout kernel execution
    int dropoutThreads = 256;
    int dropoutBlocks = (pool_output_size + dropoutThreads - 1) / dropoutThreads;
    hipEvent_t dropout_start, dropout_end;
    hipEventCreate(&dropout_start);
    hipEventCreate(&dropout_end);
    hipEventRecord(dropout_start, 0);
    dropout_forward<<<dropoutBlocks, dropoutThreads>>>(d_bn_output, d_dropout_output, pool_output_size, dropout_prob, dropout_seed);
    hipDeviceSynchronize();
    hipEventRecord(dropout_end, 0);
    hipEventSynchronize(dropout_end);
    float t_dropout = 0.0f;
    hipEventElapsedTime(&t_dropout, dropout_start, dropout_end);
    hipEventDestroy(dropout_start);
    hipEventDestroy(dropout_end);

    // Copy inference output from convolution and new layers (Device -> Host)
    float* h_gpu_dropout_output = (float*)malloc(pool_output_size * sizeof(float));
    hipEvent_t inf_start_d2h, inf_end_d2h;
    hipEventCreate(&inf_start_d2h);
    hipEventCreate(&inf_end_d2h);
    hipEventRecord(inf_start_d2h, 0);
    hipMemcpy(h_gpu_dropout_output, d_dropout_output, pool_output_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(inf_end_d2h, 0);
    hipEventSynchronize(inf_end_d2h);
    float inf_d2h_time = 0.0f;
    hipEventElapsedTime(&inf_d2h_time, inf_start_d2h, inf_end_d2h);
    hipEventDestroy(inf_start_d2h);
    hipEventDestroy(inf_end_d2h);

    float total_gpu_inf_time = inf_h2d_time + inf_kernel_time + t_pool + t_bn + t_dropout + inf_d2h_time;

    printf("\n--- Timing Results ---\n");
    printf("CPU Training: Total Epoch Time (sum): %.4f ms\n", total_cpu_epoch_time);
    printf("CPU Training: Wall Clock Time: %f ms\n", total_cpu_train_time);
    printf("CPU Inference Time (including new layers): %.4f ms\n", cpu_inf_time);
    
    printf("\n--- GPU Training Timings (convolution only) ---\n");
    printf("Total Host to Device (training): %f ms\n", total_gpu_h2d_time);
    printf("Total Kernel Execution (training): %f ms\n", total_gpu_kernel_time);
    printf("Total Device to Host (training): %f ms\n", total_gpu_d2h_time);
    printf("Total GPU Training Time: %f ms\n", total_gpu_train_time);
    
    printf("\n--- GPU Inference Timings ---\n");
    printf("Host to Device: %f ms\n", inf_h2d_time);
    printf("Convolution Kernel Execution: %f ms\n", inf_kernel_time);
    printf("Pooling Kernel: %f ms\n", t_pool);
    printf("Batch Norm Kernel: %f ms\n", t_bn);
    printf("Dropout Kernel: %f ms\n", t_dropout);
    printf("Device to Host: %f ms\n", inf_d2h_time);
    printf("Total GPU Inference Time (with new layers): %f ms\n", total_gpu_inf_time);

    // Clean up host memory
    free(h_input);
    free(h_weight);
    free(h_output);
    free(h_d_output);
    free(cpu_weight);
    free(h_pool_output);
    free(h_bn_output);
    free(h_dropout_output);
    free(h_gamma);
    free(h_beta);
    free(h_gpu_dropout_output);

    // Clean up device memory
    hipFree(d_input);
    hipFree(d_weight);
    hipFree(d_output);
    hipFree(d_d_output);
    hipFree(d_d_weight);
    hipFree(d_pool_output);
    hipFree(d_bn_output);
    hipFree(d_dropout_output);
    hipFree(d_gamma);
    hipFree(d_beta);

    return 0;
}

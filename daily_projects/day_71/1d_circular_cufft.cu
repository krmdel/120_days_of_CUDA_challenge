#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <vector>
#include <chrono>

#define CHECK_CUDA(call)                                                  \
    {                                                                     \
        hipError_t err = call;                                           \
        if (err != hipSuccess) {                                         \
            std::cerr << "CUDA error: " << hipGetErrorString(err)       \
                      << " at line " << __LINE__ << std::endl;           \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    }

#define CHECK_CUFFT(call)                                                 \
    {                                                                     \
        hipfftResult err = call;                                           \
        if (err != HIPFFT_SUCCESS) {                                       \
            std::cerr << "CUFFT error: " << err << " at line " << __LINE__ << std::endl; \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    }

// Element-wise complex multiplication kernel
__global__ void complexPointwiseMul(hipfftComplex* a, const hipfftComplex* b, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        a[i] = hipCmulf(a[i], b[i]);
    }
}

int main() {
    const int N = 16'384;
    size_t size = sizeof(hipfftComplex) * N;
    std::cout << "Vector length: " << N << "\n";

    // Host inputs
    std::vector<float> x(N, 0.0f), h(N, 0.0f);
    for (int i = 0; i < N; ++i) {
        x[i] = sinf(2 * M_PI * i / N);
        h[i] = cosf(2 * M_PI * i / N);
    }

    // Device buffers
    hipfftComplex *d_x, *d_h;
    CHECK_CUDA(hipMalloc(&d_x, size));
    CHECK_CUDA(hipMalloc(&d_h, size));

    // Copy real input as complex
    std::vector<hipfftComplex> x_c(N), h_c(N);
    for (int i = 0; i < N; ++i) {
        x_c[i] = make_hipFloatComplex(x[i], 0.0f);
        h_c[i] = make_hipFloatComplex(h[i], 0.0f);
    }

    CHECK_CUDA(hipMemcpy(d_x, x_c.data(), size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_h, h_c.data(), size, hipMemcpyHostToDevice));

    hipfftHandle plan;
    CHECK_CUFFT(hipfftPlan1d(&plan, N, HIPFFT_C2C, 1));

    hipEvent_t start, stop;
    float elapsed;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Forward FFT
    CHECK_CUFFT(hipfftExecC2C(plan, d_x, d_x, HIPFFT_FORWARD));
    CHECK_CUFFT(hipfftExecC2C(plan, d_h, d_h, HIPFFT_FORWARD));

    // Pointwise multiplication
    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    complexPointwiseMul<<<blocks, threads>>>(d_x, d_h, N);

    // Inverse FFT
    CHECK_CUFFT(hipfftExecC2C(plan, d_x, d_x, HIPFFT_BACKWARD));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);

    // Copy back result and normalize
    std::vector<hipfftComplex> result(N);
    CHECK_CUDA(hipMemcpy(result.data(), d_x, size, hipMemcpyDeviceToHost));
    for (int i = 0; i < N; ++i) {
        result[i].x /= N;
        result[i].y /= N;
    }

    // Printing inference time
    std::cout << "\nInference time: " << elapsed << " ms\n";

    // Cleanup
    hipFree(d_x);
    hipFree(d_h);
    hipfftDestroy(plan);
    return 0;
}
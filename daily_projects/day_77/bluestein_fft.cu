#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <vector>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <random>
#include <iostream>

#ifndef BLU_LEN
#define BLU_LEN 16384
#endif
static_assert(BLU_LEN > 2, "length must be >2");

// CUDA helpers
#define CUDA_CHECK(x) \
    do { hipError_t rc=(x); if(rc!=hipSuccess){                             \
        fprintf(stderr,"CUDA %s @ %s:%d\n", hipGetErrorString(rc),           \
                __FILE__,__LINE__); std::exit(EXIT_FAILURE);} } while(0)
#define CUFFT_CHECK(x) \
    do { hipfftResult rc=(x); if(rc!=HIPFFT_SUCCESS){                           \
        fprintf(stderr,"cuFFT error %d @ %s:%d\n",rc,__FILE__,__LINE__);      \
        std::exit(EXIT_FAILURE);} } while(0)

__device__ __forceinline__ hipfftComplex cmul(hipfftComplex a, hipfftComplex b){
    hipfftComplex r; r.x=a.x*b.x - a.y*b.y; r.y=a.x*b.y + a.y*b.x; return r;
}
__device__ __forceinline__ hipfftComplex cexpf_wrap(float theta){
    hipfftComplex r; sincosf(theta,&r.y,&r.x); return r;       // cos+jsin
}

// Kernel 1 – build chirp-weighted a[n] and b[n] (0≤n<N)
__global__ void build_chirp(const hipfftComplex* x,
                            hipfftComplex*       A,
                            hipfftComplex*       B,
                            int                 Nlen,
                            float               pi_over_N)
{
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if(n >= Nlen) return;
    float ang   =  pi_over_N * n * n;         //  π n² / N
    hipfftComplex w_n   = cexpf_wrap( ang);
    hipfftComplex w_n_m = cexpf_wrap(-ang);

    A[n] = cmul(x[n], w_n);                   // a[n] = x[n]·w^{ n² }
    B[n] = w_n_m;                             // b[n] =          w^{−n²}
}

//  Kernel 2 – zero-pad B to length M and copy mirrored tail
__global__ void pad_B(hipfftComplex* B, int Nlen, int Mlen){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i >= Mlen) return;

    if(i >= Nlen && i < Mlen - Nlen + 1) {        // middle region → 0
        B[i].x = B[i].y = 0.0f;
    }
    // Mirror the first (N-1) non-zero taps into the tail */
    if(i > 0 && i < Nlen){
        B[Mlen - i] = B[i];
    }
}

// Kernel 3 – element-wise complex product
__global__ void pointwise_mul(const hipfftComplex* A,
                              const hipfftComplex* B,
                              hipfftComplex*       C,
                              int                 Mlen)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < Mlen) C[i] = cmul(A[i], B[i]);
}

// Kernel 4 – final chirp & IFFT scaling
__global__ void final_chirp(const hipfftComplex* y,
                            hipfftComplex*       X,
                            int                 Nlen,
                            int                 Mlen,
                            float               pi_over_N)
{
    int k = blockIdx.x*blockDim.x + threadIdx.x;
    if(k >= Nlen) return;
    float ang = -pi_over_N * k * k;           // −π k² / N
    hipfftComplex w_k = cexpf_wrap(ang);
    hipfftComplex val = y[k];
    val.x /= Mlen;  val.y /= Mlen;            // normalise IFFT
    X[k]   = cmul(val, w_k);                  // X[k] = w^{−k²}·y[k]/M
}

// Host utility – next power-of-two
int next_pow2(int n){ int p=1; while(p<n) p<<=1; return p; }

int main()
{
    const int N = BLU_LEN;
    const int M = next_pow2(2*N - 1);         // convolution length
    const float pi_over_N = M_PI / N;

    std::cout << "Bluestein length N : " << N << "\n"
              << "Convolution M     : " << M << "\n\n";

    // Generate random input
    std::vector<hipfftComplex> h_x(N);
    std::mt19937 rng(123);
    std::uniform_real_distribution<float> dist(-1.f, 1.f);
    for(auto& v : h_x){ v.x = dist(rng); v.y = dist(rng); }

    // Device buffers
    hipfftComplex *d_x, *d_A, *d_B, *d_C, *d_X;
    CUDA_CHECK(hipMalloc(&d_x, sizeof(hipfftComplex)*N));
    CUDA_CHECK(hipMalloc(&d_A, sizeof(hipfftComplex)*M));
    CUDA_CHECK(hipMalloc(&d_B, sizeof(hipfftComplex)*M));
    CUDA_CHECK(hipMalloc(&d_C, sizeof(hipfftComplex)*M));
    CUDA_CHECK(hipMalloc(&d_X, sizeof(hipfftComplex)*N));

    // Events for timing
    hipEvent_t e0,e1,e2,e3,e4,e5;
    hipEventCreate(&e0); hipEventCreate(&e1); hipEventCreate(&e2);
    hipEventCreate(&e3); hipEventCreate(&e4); hipEventCreate(&e5);

    // H2D copy
    hipEventRecord(e0);
    CUDA_CHECK(hipMemcpy(d_x, h_x.data(),
                          sizeof(hipfftComplex)*N, hipMemcpyHostToDevice));
    hipEventRecord(e1);

    // Chirp build & padding
    int threads=256;
    int blocksN=(N+threads-1)/threads;
    int blocksM=(M+threads-1)/threads;

    build_chirp<<<blocksN,threads>>>(d_x, d_A, d_B, N, pi_over_N);
    pad_B     <<<blocksM,threads>>>(d_B, N, M);
    hipEventRecord(e2);

    // Convolution via cuFFT
    hipfftHandle plan; CUFFT_CHECK(hipfftPlan1d(&plan, M, HIPFFT_C2C, 1));

    CUFFT_CHECK(hipfftExecC2C(plan, d_A, d_A, HIPFFT_FORWARD));
    CUFFT_CHECK(hipfftExecC2C(plan, d_B, d_B, HIPFFT_FORWARD));
    pointwise_mul<<<blocksM,threads>>>(d_A, d_B, d_C, M);
    CUFFT_CHECK(hipfftExecC2C(plan, d_C, d_C, HIPFFT_BACKWARD));

    // Final chirp & scale
    final_chirp<<<blocksN,threads>>>(d_C, d_X, N, M, pi_over_N);
    hipEventRecord(e3);

    // D2H copy
    std::vector<hipfftComplex> h_X(N);
    CUDA_CHECK(hipMemcpy(h_X.data(), d_X,
                          sizeof(hipfftComplex)*N, hipMemcpyDeviceToHost));
    hipEventRecord(e4); hipEventSynchronize(e4);

    // Timings
    float tH2D, tKernel, tD2H;
    hipEventElapsedTime(&tH2D, e0, e1);
    hipEventElapsedTime(&tKernel, e1, e3);
    hipEventElapsedTime(&tD2H, e3, e4);

    std::cout << "Host → Device copy      : " << tH2D    << " ms\n";
    std::cout << "GPU kernels + FFTs      : " << tKernel << " ms\n";
    std::cout << "Device → Host copy      : " << tD2H    << " ms\n";
    std::cout << "Total GPU time          : " << tH2D + tKernel + tD2H
              << " ms\n";

    // Cleanup
    hipfftDestroy(plan);
    hipFree(d_x); hipFree(d_A); hipFree(d_B); hipFree(d_C); hipFree(d_X);
    return 0;
}
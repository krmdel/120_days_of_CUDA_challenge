#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <vector>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <random>
#include <iostream>

#ifndef N
#define N 257                    // default prime length (must be prime)
#endif
static_assert(N > 2,  "N must be > 2");
static_assert(N % 2 != 0, "N must be odd (prime)");

// CUDA helpers
#define CUDA_CHECK(x) \
    do { hipError_t rc=(x); if(rc!=hipSuccess){                              \
        std::fprintf(stderr,"CUDA %s @ %s:%d\n",hipGetErrorString(rc),        \
                     __FILE__,__LINE__); std::exit(EXIT_FAILURE);} } while(0)
#define CUFFT_CHECK(x) \
    do { hipfftResult rc=(x); if(rc!=HIPFFT_SUCCESS){                            \
        std::fprintf(stderr,"cuFFT error %d @ %s:%d\n",rc,__FILE__,__LINE__);  \
        std::exit(EXIT_FAILURE);} } while(0)

__device__ __forceinline__ float2 operator+(float2 a,float2 b){
    return make_float2(a.x+b.x , a.y+b.y);
}
__device__ __forceinline__ float2 operator*(float2 a,float2 b){
    return make_float2(a.x*b.x - a.y*b.y , a.x*b.y + a.y*b.x);
}

// Pointwise multiply kernel
__global__ void mul_complex(const hipfftComplex* A,
                            const hipfftComplex* B,
                            hipfftComplex*       C,
                            int                 n)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    if(i<n){
        hipfftComplex a=A[i], b=B[i];
        C[i].x = a.x*b.x - a.y*b.y;
        C[i].y = a.x*b.y + a.y*b.x;
    }
}

// Primitive root (host)
/*  Finds the smallest generator of (Z_N)*.  N is prime and small
( < 10^6 here ), so simple trial works.                          */
int primitive_root(int p)
{
    std::vector<int> fact;
    int phi=p-1, n=phi;
    for(int i=2; i*i<=n; ++i)
        if(n%i==0){
            fact.push_back(i);
            while(n%i==0) n/=i;
        }
    if(n>1) fact.push_back(n);
    for(int g=2; g<p; ++g){
        bool ok=true;
        for(int f:fact)
            if(std::pow(g,phi/f) - std::floor(std::pow(g,phi/f)+0.5) == 0){ // integer pow
                int t=1;
                for(int i=0;i<phi/f;++i) t=(t*g)%p;
                if(t==1){ ok=false; break;}
            }
        if(ok) return g;
    }
    return -1;
}

// Host utility: next pow-2
int next_pow2(int n){ int p=1; while(p<n) p<<=1; return p; }

int main()
{
    constexpr int PRIME = N;
    const int M  = PRIME - 1;                 // convolution length
    const int L  = next_pow2(M);              // cuFFT size
    const float TWO_PI = 2.0f * M_PI;

    std::cout<<"Prime length N  : "<<PRIME<<"\n";
    std::cout<<"Convolution len : "<<M<<"\n";
    std::cout<<"FFT size (pow2) : "<<L<<"\n\n";

    // Generate random complex input x[n]
    std::vector<float2> h_x(PRIME);
    std::mt19937 rng(42);
    std::uniform_real_distribution<float> dist(-1.f,1.f);
    for(auto& v: h_x){ v.x=dist(rng); v.y=dist(rng); }

    // Step 1 : choose primitive root g
    int g = 3;                       // 3 is a primitive root for many primes incl. 257
#if N != 257
    g = primitive_root(PRIME);
#endif
    if(g<0){ std::cerr<<"Cannot find primitive root\n"; return 1; }

    // Step 2 : build sequences A (re-ordered data) and B (twiddles)
    std::vector<hipfftComplex> A(L), B(L);
    int k=1;
    for(int m=0;m<M;++m){
        A[m].x = h_x[k].x;  A[m].y = h_x[k].y;          // a_m = x[g^m]
        float ang = -TWO_PI * k / PRIME;
        B[m].x = cosf(ang);  B[m].y = sinf(ang);        // b_m = e^{-j2π k/N}
        k = (k * g) % PRIME;
    }
    for(int i=M;i<L;++i){ A[i].x=A[i].y=0; B[i].x=B[i].y=0; }

    // GPU buffers
    hipfftComplex *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc(&d_A,sizeof(hipfftComplex)*L));
    CUDA_CHECK(hipMalloc(&d_B,sizeof(hipfftComplex)*L));
    CUDA_CHECK(hipMalloc(&d_C,sizeof(hipfftComplex)*L));

    // Events for timing
    hipEvent_t e0,e1,e2,e3,e4; hipEventCreate(&e0);hipEventCreate(&e1);
    hipEventCreate(&e2);hipEventCreate(&e3);hipEventCreate(&e4);

    hipEventRecord(e0);
    CUDA_CHECK(hipMemcpy(d_A,A.data(),sizeof(hipfftComplex)*L,hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B,B.data(),sizeof(hipfftComplex)*L,hipMemcpyHostToDevice));
    hipEventRecord(e1);

    // cuFFT plans
    hipfftHandle planF, planI;
    CUFFT_CHECK(hipfftPlan1d(&planF,L,HIPFFT_C2C,1));
    CUFFT_CHECK(hipfftPlan1d(&planI,L,HIPFFT_C2C,1));

    // Forward FFT
    CUFFT_CHECK(hipfftExecC2C(planF,d_A,d_A,HIPFFT_FORWARD));
    CUFFT_CHECK(hipfftExecC2C(planF,d_B,d_B,HIPFFT_FORWARD));

    // Pointwise multiply
    int threads=256, blocks=(L+threads-1)/threads;
    mul_complex<<<blocks,threads>>>(d_A,d_B,d_C,L);

    // Inverse FFT
    CUFFT_CHECK(hipfftExecC2C(planI,d_C,d_C,HIPFFT_BACKWARD));
    hipEventRecord(e2);

    // Copy convolution result back
    std::vector<hipfftComplex> h_C(L);
    CUDA_CHECK(hipMemcpy(h_C.data(),d_C,sizeof(hipfftComplex)*L,hipMemcpyDeviceToHost));
    hipEventRecord(e3); hipEventSynchronize(e3);

    CUDA_CHECK(hipFree(d_A)); CUDA_CHECK(hipFree(d_B)); CUDA_CHECK(hipFree(d_C));
    hipfftDestroy(planF); hipfftDestroy(planI);

    // Build final DFT result X[k]
    std::vector<float2> X(PRIME);
    float2 X0{0,0};
    for(auto &v: h_x){ X0.x+=v.x; X0.y+=v.y; }
    X[0]=X0;

    // Normalise IFFT output (cuFFT does not scale)
    for(int i=0;i<M;++i){ h_C[i].x/=L; h_C[i].y/=L; }

    int idx=1;
    for(int m=0;m<M;++m){
        X[idx].x = X0.x + h_C[m].x;
        X[idx].y = X0.y + h_C[m].y;
        idx = (idx * g) % PRIME;
    }

    hipEventRecord(e4); hipEventSynchronize(e4);

    // Timings
    float tH2D,tFFT,tMul,tD2H;
    hipEventElapsedTime(&tH2D,e0,e1);
    hipEventElapsedTime(&tFFT,e1,e2);              // includes point-mult
    hipEventElapsedTime(&tMul,e2,e3);              // IFFT+copy
    hipEventElapsedTime(&tD2H,e3,e4);

    std::cout<<"H2D copy        : "<<tH2D<<" ms\n";
    std::cout<<"FFT+mul+IFFT    : "<<tFFT<<" ms\n";
    std::cout<<"D2H copy+build  : "<<tMul+tD2H<<" ms\n";
    std::cout<<"Total GPU time  : "<<tH2D+tFFT+tMul+tD2H<<" ms\n";

    return 0;
}
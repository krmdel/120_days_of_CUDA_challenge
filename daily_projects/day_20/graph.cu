#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define INF 0x3f3f3f3f
#define BLOCK_SIZE 256

// Macro to check for CUDA errors
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if(code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if(abort) exit(code);
   }
}

__global__ void bfs_top_down_kernel(int V, const int *row_offsets, const int *col_indices,
                                      int current_level, int *distances, const int *frontier, int frontier_size,
                                      int *next_frontier, int *next_frontier_size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < frontier_size) {
        int u = frontier[tid];
        int row_start = row_offsets[u];
        int row_end = row_offsets[u+1];
        for (int offset = row_start; offset < row_end; offset++) {
            int v = col_indices[offset];
            if (atomicCAS(&distances[v], INF, current_level + 1) == INF) {
                int pos = atomicAdd(next_frontier_size, 1);
                next_frontier[pos] = v;
            }
        }
    }
}

__global__ void bfs_bottom_up_kernel(int V, const int *row_offsets, const int *col_indices,
                                       int current_level, int *distances, int *changed) {
    int u = blockIdx.x * blockDim.x + threadIdx.x;
    if(u < V && distances[u] == INF) {
        int row_start = row_offsets[u];
        int row_end = row_offsets[u+1];
        for (int offset = row_start; offset < row_end; offset++) {
            int v = col_indices[offset];
            if(distances[v] == current_level) {
                distances[u] = current_level + 1;
                *changed = 1;
                break;
            }
        }
    }
}

__global__ void bfs_edge_centric_kernel(int E, const int *edge_src, const int *edge_dst,
                                          int current_level, int *distances, int *changed) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < E) {
        int u = edge_src[tid];
        int v = edge_dst[tid];
        if(distances[u] == current_level && distances[v] == INF) {
            if(atomicCAS(&distances[v], INF, current_level + 1) == INF) {
                *changed = 1;
            }
        }
        if(distances[v] == current_level && distances[u] == INF) {
            if(atomicCAS(&distances[u], INF, current_level + 1) == INF) {
                *changed = 1;
            }
        }
    }
}

__global__ void count_frontier_kernel(int V, const int *distances, int level, int *frontier_count) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < V) {
        if(distances[tid] == level + 1)
            atomicAdd(frontier_count, 1);
    }
}

void run_bfs_top_down(const int *h_row_offsets, const int *h_col_indices, int V, int E, int source) {

    // Create CUDA events for timing.
    hipEvent_t start, copyHtoD, kernelStart, kernelEnd, copyDtoH, end;
    hipEventCreate(&start);
    hipEventCreate(&copyHtoD);
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelEnd);
    hipEventCreate(&copyDtoH);
    hipEventCreate(&end);

    hipEventRecord(start, 0);

    // Allocate and copy graph data to device
    int *d_row_offsets, *d_col_indices;
    gpuErrchk( hipMalloc((void**)&d_row_offsets, sizeof(int)*(V+1)) );
    gpuErrchk( hipMalloc((void**)&d_col_indices, sizeof(int)*E) );
    gpuErrchk( hipMemcpy(d_row_offsets, h_row_offsets, sizeof(int)*(V+1), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_col_indices, h_col_indices, sizeof(int)*E, hipMemcpyHostToDevice) );

    // Allocate distances array on device
    int *d_distances;
    gpuErrchk( hipMalloc((void**)&d_distances, sizeof(int)*V) );
    gpuErrchk( hipMemset(d_distances, 0x3f, sizeof(int)*V) );
    int zero = 0;
    gpuErrchk( hipMemcpy(&d_distances[source], &zero, sizeof(int), hipMemcpyHostToDevice) );

    // Allocate frontier arrays on device
    int *d_frontier_current, *d_frontier_next;
    gpuErrchk( hipMalloc((void**)&d_frontier_current, sizeof(int)*V) );
    gpuErrchk( hipMalloc((void**)&d_frontier_next, sizeof(int)*V) );
    gpuErrchk( hipMemcpy(d_frontier_current, &source, sizeof(int), hipMemcpyHostToDevice) );
    int frontier_size = 1;

    int *d_frontier_size;
    gpuErrchk( hipMalloc((void**)&d_frontier_size, sizeof(int)) );

    hipEventRecord(copyHtoD, 0);

    // Begin BFS
    int level = 0;
    hipEventRecord(kernelStart, 0);
    while(frontier_size > 0) {
        gpuErrchk( hipMemset(d_frontier_size, 0, sizeof(int)) );

        int num_blocks = (frontier_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
        bfs_top_down_kernel<<<num_blocks, BLOCK_SIZE>>>(V, d_row_offsets, d_col_indices,
                                                          level, d_distances, d_frontier_current, frontier_size,
                                                          d_frontier_next, d_frontier_size);
        gpuErrchk( hipDeviceSynchronize() );

        int new_frontier_size;
        gpuErrchk( hipMemcpy(&new_frontier_size, d_frontier_size, sizeof(int), hipMemcpyDeviceToHost) );
        frontier_size = new_frontier_size;

        int *temp = d_frontier_current;
        d_frontier_current = d_frontier_next;
        d_frontier_next = temp;
        level++;
    }
    hipEventRecord(kernelEnd, 0);

    int *h_distances = (int*)malloc(sizeof(int)*V);
    gpuErrchk( hipMemcpy(h_distances, d_distances, sizeof(int)*V, hipMemcpyDeviceToHost) );
    hipEventRecord(copyDtoH, 0);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    float time_copyHtoD, time_kernel, time_copyDtoH, time_total;
    hipEventElapsedTime(&time_copyHtoD, start, copyHtoD);
    hipEventElapsedTime(&time_kernel, kernelStart, kernelEnd);
    hipEventElapsedTime(&time_copyDtoH, copyDtoH, end);
    hipEventElapsedTime(&time_total, start, end);

    printf("Top-Down BFS Timings (ms): Copy H->D: %f, Kernel: %f, Copy D->H: %f, Total: %f\n",
           time_copyHtoD, time_kernel, time_copyDtoH, time_total);

    hipFree(d_row_offsets);
    hipFree(d_col_indices);
    hipFree(d_distances);
    hipFree(d_frontier_current);
    hipFree(d_frontier_next);
    hipFree(d_frontier_size);
    free(h_distances);

    hipEventDestroy(start);
    hipEventDestroy(copyHtoD);
    hipEventDestroy(kernelStart);
    hipEventDestroy(kernelEnd);
    hipEventDestroy(copyDtoH);
    hipEventDestroy(end);
}

void run_bfs_bottom_up(const int *h_row_offsets, const int *h_col_indices, int V, int source) {

    hipEvent_t start, copyHtoD, kernelStart, kernelEnd, copyDtoH, end;
    hipEventCreate(&start);
    hipEventCreate(&copyHtoD);
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelEnd);
    hipEventCreate(&copyDtoH);
    hipEventCreate(&end);

    hipEventRecord(start, 0);

    // Copy CSR graph to device
    int *d_row_offsets, *d_col_indices;
    int E = h_row_offsets[V];
    gpuErrchk( hipMalloc((void**)&d_row_offsets, sizeof(int)*(V+1)) );
    gpuErrchk( hipMalloc((void**)&d_col_indices, sizeof(int)*E) );
    gpuErrchk( hipMemcpy(d_row_offsets, h_row_offsets, sizeof(int)*(V+1), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_col_indices, h_col_indices, sizeof(int)*E, hipMemcpyHostToDevice) );

    // Allocate and initialize distances
    int *d_distances;
    gpuErrchk( hipMalloc((void**)&d_distances, sizeof(int)*V) );
    gpuErrchk( hipMemset(d_distances, 0x3f, sizeof(int)*V) );
    int zero = 0;
    gpuErrchk( hipMemcpy(&d_distances[source], &zero, sizeof(int), hipMemcpyHostToDevice) );

    hipEventRecord(copyHtoD, 0);

    int level = 0;
    int h_changed;
    int *d_changed;
    gpuErrchk( hipMalloc((void**)&d_changed, sizeof(int)) );

    hipEventRecord(kernelStart, 0);
    do {
        h_changed = 0;
        gpuErrchk( hipMemcpy(d_changed, &h_changed, sizeof(int), hipMemcpyHostToDevice) );
        int num_blocks = (V + BLOCK_SIZE - 1) / BLOCK_SIZE;
        bfs_bottom_up_kernel<<<num_blocks, BLOCK_SIZE>>>(V, d_row_offsets, d_col_indices, level, d_distances, d_changed);
        gpuErrchk( hipDeviceSynchronize() );
        gpuErrchk( hipMemcpy(&h_changed, d_changed, sizeof(int), hipMemcpyDeviceToHost) );
        level++;
    } while(h_changed);
    hipEventRecord(kernelEnd, 0);

    // Copy distances back to host
    int *h_distances = (int*)malloc(sizeof(int)*V);
    gpuErrchk( hipMemcpy(h_distances, d_distances, sizeof(int)*V, hipMemcpyDeviceToHost) );
    hipEventRecord(copyDtoH, 0);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    float time_copyHtoD, time_kernel, time_copyDtoH, time_total;
    hipEventElapsedTime(&time_copyHtoD, start, copyHtoD);
    hipEventElapsedTime(&time_kernel, kernelStart, kernelEnd);
    hipEventElapsedTime(&time_copyDtoH, copyDtoH, end);
    hipEventElapsedTime(&time_total, start, end);

    printf("Bottom-Up BFS Timings (ms): Copy H->D: %f, Kernel: %f, Copy D->H: %f, Total: %f\n",
           time_copyHtoD, time_kernel, time_copyDtoH, time_total);

    hipFree(d_row_offsets);
    hipFree(d_col_indices);
    hipFree(d_distances);
    hipFree(d_changed);
    free(h_distances);

    hipEventDestroy(start);
    hipEventDestroy(copyHtoD);
    hipEventDestroy(kernelStart);
    hipEventDestroy(kernelEnd);
    hipEventDestroy(copyDtoH);
    hipEventDestroy(end);
}

void run_bfs_direction_optimized(const int *h_row_offsets, const int *h_col_indices, int V, int source) {

    hipEvent_t start, copyHtoD, kernelStart, kernelMiddle, kernelEnd, copyDtoH, end;
    hipEventCreate(&start);
    hipEventCreate(&copyHtoD);
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelMiddle);
    hipEventCreate(&kernelEnd);
    hipEventCreate(&copyDtoH);
    hipEventCreate(&end);

    hipEventRecord(start, 0);

    // Copy graph data
    int *d_row_offsets, *d_col_indices;
    int E = h_row_offsets[V];
    gpuErrchk( hipMalloc((void**)&d_row_offsets, sizeof(int)*(V+1)) );
    gpuErrchk( hipMalloc((void**)&d_col_indices, sizeof(int)*E) );
    gpuErrchk( hipMemcpy(d_row_offsets, h_row_offsets, sizeof(int)*(V+1), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_col_indices, h_col_indices, sizeof(int)*E, hipMemcpyHostToDevice) );

    // Allocate and initialize distances
    int *d_distances;
    gpuErrchk( hipMalloc((void**)&d_distances, sizeof(int)*V) );
    gpuErrchk( hipMemset(d_distances, 0x3f, sizeof(int)*V) );
    int zero = 0;
    gpuErrchk( hipMemcpy(&d_distances[source], &zero, sizeof(int), hipMemcpyHostToDevice) );

    // Allocate frontier arrays
    int *d_frontier_current, *d_frontier_next;
    gpuErrchk( hipMalloc((void**)&d_frontier_current, sizeof(int)*V) );
    gpuErrchk( hipMalloc((void**)&d_frontier_next, sizeof(int)*V) );
    gpuErrchk( hipMemcpy(d_frontier_current, &source, sizeof(int), hipMemcpyHostToDevice) );
    int frontier_size = 1;

    int *d_frontier_size;
    gpuErrchk( hipMalloc((void**)&d_frontier_size, sizeof(int)) );

    hipEventRecord(copyHtoD, 0);

    int level = 0;
    const int threshold = V / 10;
    hipEventRecord(kernelStart, 0);
    // Top-down phase
    while(frontier_size > 0 && frontier_size < threshold) {
        gpuErrchk( hipMemset(d_frontier_size, 0, sizeof(int)) );
        int num_blocks = (frontier_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
        bfs_top_down_kernel<<<num_blocks, BLOCK_SIZE>>>(V, d_row_offsets, d_col_indices,
                                                          level, d_distances, d_frontier_current, frontier_size,
                                                          d_frontier_next, d_frontier_size);
        gpuErrchk( hipDeviceSynchronize() );
        int new_frontier_size;
        gpuErrchk( hipMemcpy(&new_frontier_size, d_frontier_size, sizeof(int), hipMemcpyDeviceToHost) );
        frontier_size = new_frontier_size;
        int *temp = d_frontier_current;
        d_frontier_current = d_frontier_next;
        d_frontier_next = temp;
        level++;
    }
    hipEventRecord(kernelMiddle, 0);
    // Bottom-up phase
    int h_changed;
    int *d_changed;
    gpuErrchk( hipMalloc((void**)&d_changed, sizeof(int)) );
    while(true) {
        h_changed = 0;
        gpuErrchk( hipMemcpy(d_changed, &h_changed, sizeof(int), hipMemcpyHostToDevice) );
        int num_blocks = (V + BLOCK_SIZE - 1) / BLOCK_SIZE;
        bfs_bottom_up_kernel<<<num_blocks, BLOCK_SIZE>>>(V, d_row_offsets, d_col_indices, level, d_distances, d_changed);
        gpuErrchk( hipDeviceSynchronize() );
        gpuErrchk( hipMemcpy(&h_changed, d_changed, sizeof(int), hipMemcpyDeviceToHost) );
        if(h_changed == 0) break;
        level++;
    }
    hipEventRecord(kernelEnd, 0);

    // Copy distances back to host
    int *h_distances = (int*)malloc(sizeof(int)*V);
    gpuErrchk( hipMemcpy(h_distances, d_distances, sizeof(int)*V, hipMemcpyDeviceToHost) );
    hipEventRecord(copyDtoH, 0);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    float time_copyHtoD, time_kernel, time_copyDtoH, time_total;
    hipEventElapsedTime(&time_copyHtoD, start, copyHtoD);
    hipEventElapsedTime(&time_kernel, kernelStart, kernelEnd);
    hipEventElapsedTime(&time_copyDtoH, copyDtoH, end);
    hipEventElapsedTime(&time_total, start, end);

    printf("Direction-Optimized BFS Timings (ms): Copy H->D: %f, Kernel: %f, Copy D->H: %f, Total: %f\n",
           time_copyHtoD, time_kernel, time_copyDtoH, time_total);

    hipFree(d_row_offsets);
    hipFree(d_col_indices);
    hipFree(d_distances);
    hipFree(d_frontier_current);
    hipFree(d_frontier_next);
    hipFree(d_frontier_size);
    hipFree(d_changed);
    free(h_distances);

    hipEventDestroy(start);
    hipEventDestroy(copyHtoD);
    hipEventDestroy(kernelStart);
    hipEventDestroy(kernelMiddle);
    hipEventDestroy(kernelEnd);
    hipEventDestroy(copyDtoH);
    hipEventDestroy(end);
}

void run_bfs_edge_centric(const int *h_src, const int *h_dst, int V, int E, int source) {

    hipEvent_t start, copyHtoD, kernelStart, kernelEnd, copyDtoH, end;
    hipEventCreate(&start);
    hipEventCreate(&copyHtoD);
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelEnd);
    hipEventCreate(&copyDtoH);
    hipEventCreate(&end);

    hipEventRecord(start, 0);

    // Copy COO graph to device
    int *d_edge_src, *d_edge_dst;
    gpuErrchk( hipMalloc((void**)&d_edge_src, sizeof(int)*E) );
    gpuErrchk( hipMalloc((void**)&d_edge_dst, sizeof(int)*E) );
    gpuErrchk( hipMemcpy(d_edge_src, h_src, sizeof(int)*E, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_edge_dst, h_dst, sizeof(int)*E, hipMemcpyHostToDevice) );

    // Allocate and initialize distances
    int *d_distances;
    gpuErrchk( hipMalloc((void**)&d_distances, sizeof(int)*V) );
    gpuErrchk( hipMemset(d_distances, 0x3f, sizeof(int)*V) );
    int zero = 0;
    gpuErrchk( hipMemcpy(&d_distances[source], &zero, sizeof(int), hipMemcpyHostToDevice) );

    hipEventRecord(copyHtoD, 0);

    int level = 0;
    int h_changed;
    int *d_changed;
    gpuErrchk( hipMalloc((void**)&d_changed, sizeof(int)) );

    hipEventRecord(kernelStart, 0);
    while(true) {
        h_changed = 0;
        gpuErrchk( hipMemcpy(d_changed, &h_changed, sizeof(int), hipMemcpyHostToDevice) );
        int num_blocks = (E + BLOCK_SIZE - 1) / BLOCK_SIZE;
        bfs_edge_centric_kernel<<<num_blocks, BLOCK_SIZE>>>(E, d_edge_src, d_edge_dst, level, d_distances, d_changed);
        gpuErrchk( hipDeviceSynchronize() );
        gpuErrchk( hipMemcpy(&h_changed, d_changed, sizeof(int), hipMemcpyDeviceToHost) );
        if(h_changed == 0) break;
        level++;
    }
    hipEventRecord(kernelEnd, 0);

    // Copy distances back
    int *h_distances = (int*)malloc(sizeof(int)*V);
    gpuErrchk( hipMemcpy(h_distances, d_distances, sizeof(int)*V, hipMemcpyDeviceToHost) );
    hipEventRecord(copyDtoH, 0);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    float time_copyHtoD, time_kernel, time_copyDtoH, time_total;
    hipEventElapsedTime(&time_copyHtoD, start, copyHtoD);
    hipEventElapsedTime(&time_kernel, kernelStart, kernelEnd);
    hipEventElapsedTime(&time_copyDtoH, copyDtoH, end);
    hipEventElapsedTime(&time_total, start, end);

    printf("Edge-Centric BFS Timings (ms): Copy H->D: %f, Kernel: %f, Copy D->H: %f, Total: %f\n",
           time_copyHtoD, time_kernel, time_copyDtoH, time_total);

    hipFree(d_edge_src);
    hipFree(d_edge_dst);
    hipFree(d_distances);
    hipFree(d_changed);
    free(h_distances);

    hipEventDestroy(start);
    hipEventDestroy(copyHtoD);
    hipEventDestroy(kernelStart);
    hipEventDestroy(kernelEnd);
    hipEventDestroy(copyDtoH);
    hipEventDestroy(end);
}

int main() {
    // Sample graph (undirected, unweighted)
    // 7 vertices and 12 edges in CSR format.
    // Vertex 0: neighbors 1,2  
    // Vertex 1: neighbors 0,3,4  
    // Vertex 2: neighbors 0,5  
    // Vertex 3: neighbor 1  
    // Vertex 4: neighbors 1,6  
    // Vertex 5: neighbor 2  
    // Vertex 6: neighbor 4
    int V = 7;
    int h_row_offsets[] = {0, 2, 5, 7, 8, 10, 11, 12};
    int h_col_indices[] = {1, 2, 0, 3, 4, 0, 5, 1, 1, 6, 2, 4};

    // COO representation for edge-centric BFS.
    int E = 12;
    int h_src[] = {0, 0, 1, 1, 1, 2, 2, 3, 4, 4, 5, 6};
    int h_dst[] = {1, 2, 0, 3, 4, 0, 5, 1, 1, 6, 2, 4};

    int source = 0;

    run_bfs_top_down(h_row_offsets, h_col_indices, V, h_row_offsets[V], source);
    run_bfs_bottom_up(h_row_offsets, h_col_indices, V, source);
    run_bfs_direction_optimized(h_row_offsets, h_col_indices, V, source);
    run_bfs_edge_centric(h_src, h_dst, V, E, source);

    return 0;
}

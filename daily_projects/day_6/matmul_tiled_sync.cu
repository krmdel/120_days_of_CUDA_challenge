#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define TILE_WIDTH 32

__global__ void matmul_tiled_kernel_sync(const float* A, const float* B, float* C, unsigned int N)
{
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    unsigned int bx = blockIdx.x;   
    unsigned int by = blockIdx.y;   
    unsigned int tx = threadIdx.x; 
    unsigned int ty = threadIdx.y; 

    unsigned int Row = by * TILE_WIDTH + ty;
    unsigned int Col = bx * TILE_WIDTH + tx;

    float Cvalue = 0.0f;

    unsigned int numPhases = (N + TILE_WIDTH - 1) / TILE_WIDTH;
    for (unsigned int ph = 0; ph < numPhases; ph++)
    {
        unsigned int tiledACol = ph * TILE_WIDTH + tx;
        if (Row < N && tiledACol < N) {
            ds_A[ty][tx] = A[Row*N + tiledACol];
        } else {
            ds_A[ty][tx] = 0.0f;
        }

        unsigned int tiledBRow = ph * TILE_WIDTH + ty;
        if (Col < N && tiledBRow < N) {
            ds_B[ty][tx] = B[tiledBRow*N + Col];
        } else {
            ds_B[ty][tx] = 0.0f;
        }

        __syncthreads();  // Sync threads in block

        for (int k = 0; k < TILE_WIDTH; k++) {
            Cvalue += ds_A[ty][k] * ds_B[k][tx];
        }

        __syncthreads(); // Sync threads in block
    }

    // Store result
    if (Row < N && Col < N) {
        C[Row*N + Col] = Cvalue;
    }
}

inline void checkCuda(hipError_t result, const char* msg)
{
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s\n", msg, hipGetErrorString(result));
        exit(EXIT_FAILURE);
    }
}

void matmul_gpu_tiled_base(const float* h_A, const float* h_B, float* h_C, unsigned int N)
{
    // Create timing events
    hipEvent_t startAlloc, stopAlloc;
    hipEvent_t startCopyH2D, stopCopyH2D;
    hipEvent_t startKernel, stopKernel;
    hipEvent_t startCopyD2H, stopCopyD2H;

    checkCuda(hipEventCreate(&startAlloc),   "EventCreate startAlloc");
    checkCuda(hipEventCreate(&stopAlloc),    "EventCreate stopAlloc");
    checkCuda(hipEventCreate(&startCopyH2D), "EventCreate startCopyH2D");
    checkCuda(hipEventCreate(&stopCopyH2D),  "EventCreate stopCopyH2D");
    checkCuda(hipEventCreate(&startKernel),  "EventCreate startKernel");
    checkCuda(hipEventCreate(&stopKernel),   "EventCreate stopKernel");
    checkCuda(hipEventCreate(&startCopyD2H), "EventCreate startCopyD2H");
    checkCuda(hipEventCreate(&stopCopyD2H),  "EventCreate stopCopyD2H");

    float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr;

    // Allocate memory on GPU
    hipEventRecord(startAlloc);
    checkCuda(hipMalloc((void**)&d_A, N*N*sizeof(float)), "malloc d_A");
    checkCuda(hipMalloc((void**)&d_B, N*N*sizeof(float)), "malloc d_B");
    checkCuda(hipMalloc((void**)&d_C, N*N*sizeof(float)), "malloc d_C");
    hipDeviceSynchronize();
    hipEventRecord(stopAlloc);
    hipEventSynchronize(stopAlloc);

    float msAlloc;
    hipEventElapsedTime(&msAlloc, startAlloc, stopAlloc);

    // Copy data from CPU to GPU
    hipEventRecord(startCopyH2D);
    checkCuda(hipMemcpy(d_A, h_A, N*N*sizeof(float), hipMemcpyHostToDevice), "memcpy A->d_A");
    checkCuda(hipMemcpy(d_B, h_B, N*N*sizeof(float), hipMemcpyHostToDevice), "memcpy B->d_B");
    hipDeviceSynchronize();
    hipEventRecord(stopCopyH2D);
    hipEventSynchronize(stopCopyH2D);

    float msH2D;
    hipEventElapsedTime(&msH2D, startCopyH2D, stopCopyH2D);

    // Perform the matrix multiplication
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
    dim3 gridDim((N + TILE_WIDTH - 1)/TILE_WIDTH, (N + TILE_WIDTH - 1)/TILE_WIDTH);

    // *** FIX: Record kernel events properly ***
    hipEventRecord(startKernel);
    matmul_tiled_kernel_sync<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    hipEventRecord(stopKernel);
    hipEventSynchronize(stopKernel);

    float msKernel;
    hipEventElapsedTime(&msKernel, startKernel, stopKernel);

    // Copy the result from GPU to CPU
    hipEventRecord(startCopyD2H);
    checkCuda(hipMemcpy(h_C, d_C, N*N*sizeof(float), hipMemcpyDeviceToHost), "memcpy d_C->C");
    hipDeviceSynchronize();
    hipEventRecord(stopCopyD2H);
    hipEventSynchronize(stopCopyD2H);

    float msD2H;
    hipEventElapsedTime(&msD2H, startCopyD2H, stopCopyD2H);

    float total = msAlloc + msH2D + msKernel + msD2H;
    printf("=== Baseline Tiled Kernel ===\n");
    printf("Alloc:           %f ms\n", msAlloc);
    printf("Host->Device:    %f ms\n", msH2D);
    printf("Kernel:          %f ms\n", msKernel);
    printf("Device->Host:    %f ms\n", msD2H);
    printf("Total:           %f ms\n\n", total);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipEventDestroy(startAlloc);
    hipEventDestroy(stopAlloc);
    hipEventDestroy(startCopyH2D);
    hipEventDestroy(stopCopyH2D);
    hipEventDestroy(startKernel);
    hipEventDestroy(stopKernel);
    hipEventDestroy(startCopyD2H);
    hipEventDestroy(stopCopyD2H);
}

void matmul_gpu_tiled_extended(const float* h_A, const float* h_B, float* h_C, unsigned int N)
{
    // Create timing events
    hipEvent_t startAlloc, stopAlloc;
    hipEvent_t startCopyH2D, stopCopyH2D;
    hipEvent_t startKernel, stopKernel;
    hipEvent_t startCopyD2H, stopCopyD2H;

    checkCuda(hipEventCreate(&startAlloc),   "EventCreate startAlloc");
    checkCuda(hipEventCreate(&stopAlloc),    "EventCreate stopAlloc");
    checkCuda(hipEventCreate(&startCopyH2D), "EventCreate startCopyH2D");
    checkCuda(hipEventCreate(&stopCopyH2D),  "EventCreate stopCopyH2D");
    checkCuda(hipEventCreate(&startKernel),  "EventCreate startKernel");
    checkCuda(hipEventCreate(&stopKernel),   "EventCreate stopKernel");
    checkCuda(hipEventCreate(&startCopyD2H), "EventCreate startCopyD2H");
    checkCuda(hipEventCreate(&stopCopyD2H),  "EventCreate stopCopyD2H");

    float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr;

    // Allocate memory on GPU
    hipEventRecord(startAlloc);
    checkCuda(hipMalloc((void**)&d_A, N*N*sizeof(float)), "malloc d_A");
    checkCuda(hipMalloc((void**)&d_B, N*N*sizeof(float)), "malloc d_B");
    checkCuda(hipMalloc((void**)&d_C, N*N*sizeof(float)), "malloc d_C");
    hipDeviceSynchronize();
    hipEventRecord(stopAlloc);
    hipEventSynchronize(stopAlloc);

    float msAlloc;
    hipEventElapsedTime(&msAlloc, startAlloc, stopAlloc);

    // Copy data from CPU to GPU
    hipEventRecord(startCopyH2D);
    checkCuda(hipMemcpy(d_A, h_A, N*N*sizeof(float), hipMemcpyHostToDevice), "memcpy A->d_A");
    checkCuda(hipMemcpy(d_B, h_B, N*N*sizeof(float), hipMemcpyHostToDevice), "memcpy B->d_B");
    hipDeviceSynchronize();
    hipEventRecord(stopCopyH2D);
    hipEventSynchronize(stopCopyH2D);

    float msH2D;
    hipEventElapsedTime(&msH2D, startCopyH2D, stopCopyH2D);

    // Perform the matrix multiplication
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
    dim3 gridDim((N + TILE_WIDTH - 1)/TILE_WIDTH, (N + TILE_WIDTH - 1)/TILE_WIDTH);

    // *** FIX: Record kernel events properly ***
    hipEventRecord(startKernel);
    matmul_tiled_kernel_sync<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    hipEventRecord(stopKernel);
    hipEventSynchronize(stopKernel);

    float msKernel;
    hipEventElapsedTime(&msKernel, startKernel, stopKernel);

    // Copy the result from GPU to CPU
    hipEventRecord(startCopyD2H);
    checkCuda(hipMemcpy(h_C, d_C, N*N*sizeof(float), hipMemcpyDeviceToHost), "memcpy d_C->C");
    hipDeviceSynchronize();
    hipEventRecord(stopCopyD2H);
    hipEventSynchronize(stopCopyD2H);

    float msD2H;
    hipEventElapsedTime(&msD2H, startCopyD2H, stopCopyD2H);

    float total = msAlloc + msH2D + msKernel + msD2H;
    printf("=== Extended Tiled Kernel (Extra sync) ===\n");
    printf("Alloc:           %f ms\n", msAlloc);
    printf("Host->Device:    %f ms\n", msH2D);
    printf("Kernel:          %f ms\n", msKernel);
    printf("Device->Host:    %f ms\n", msD2H);
    printf("Total:           %f ms\n\n", total);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipEventDestroy(startAlloc);
    hipEventDestroy(stopAlloc);
    hipEventDestroy(startCopyH2D);
    hipEventDestroy(stopCopyH2D);
    hipEventDestroy(startKernel);
    hipEventDestroy(stopKernel);
    hipEventDestroy(startCopyD2H);
    hipEventDestroy(stopCopyD2H);
}

int main(int argc, char** argv)
{
    unsigned int N = 1024;
    if (argc > 1) {
        N = atoi(argv[1]);
    }
    printf("Matrix dimension: %u x %u\n", N, N);

    // Allocate memory on CPU
    float* h_A = (float*)malloc(N*N*sizeof(float));
    float* h_B = (float*)malloc(N*N*sizeof(float));
    float* h_C = (float*)malloc(N*N*sizeof(float));

    for (unsigned int i = 0; i < N*N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Matrix multiplication with tiled kernel
    matmul_gpu_tiled_base(h_A, h_B, h_C, N);

    // Matrix multiplication with extended tiled kernel
    matmul_gpu_tiled_extended(h_A, h_B, h_C, N);

    // Cleanup
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}

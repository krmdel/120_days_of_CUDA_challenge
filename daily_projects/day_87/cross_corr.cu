#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <vector>
#include <random>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <cstdlib>
#include <algorithm>          // std::max_element

// Helpers
#define THREADS 256
#define CUDA_CHECK(x)  do{ hipError_t rc=(x); if(rc!=hipSuccess){            \
    std::cerr<<"CUDA "<<hipGetErrorString(rc)<<" @"<<__FILE__<<":"<<__LINE__;\
    std::exit(EXIT_FAILURE);} }while(0)
#define CUFFT_CHECK(x) do{ hipfftResult rc=(x); if(rc!=HIPFFT_SUCCESS){          \
    std::cerr<<"cuFFT "<<rc<<" @"<<__FILE__<<":"<<__LINE__<<"\n";              \
    std::exit(EXIT_FAILURE);} }while(0)

// Naïve O(N²) kernel (one thread per lag)
__global__ void xcorr_naive(const float* __restrict__ x,
                            const float* __restrict__ y,
                            float*       __restrict__ c,
                            int N)
{
    int M   = 2*N - 1;
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx >= M) return;

    int lag = idx - (N - 1);          // negative ⇒ y leads, positive ⇒ y lags
    float acc = 0.f;
    for(int n=0; n<N; ++n){
        int j = n + lag;
        if(j >= 0 && j < N) acc += x[n] * y[j];
    }
    c[idx] = acc;
}

// Complex multiply: X · conj(Y)
__global__ void cmul_conj(const hipfftComplex* __restrict__ A,
                          const hipfftComplex* __restrict__ B,
                          hipfftComplex*       __restrict__ C,
                          int M)
{
    int k = blockIdx.x*blockDim.x + threadIdx.x;
    if(k < M){
        hipfftComplex a = A[k], b = B[k];
        C[k].x = a.x*b.x + a.y*b.y;           // real( a * conj(b) )
        C[k].y = a.y*b.x - a.x*b.y;           // imag( … )
    }
}

// Copy first M samples of IFFT result & scale
__global__ void scaleCopyKernel(float* dst,
                                const hipfftComplex* src,
                                float s, int M)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < M) dst[i] = src[i].x * s;          // take real part
}

// Utility
static int next_pow2(int v){ int p=1; while(p<v) p<<=1; return p; }
struct Times{ float h2d, ker, d2h; float total()const{return h2d+ker+d2h;} };

int main(int argc,char**argv)
{
    const int N = (argc>1)? std::atoi(argv[1]) : 262'144;   // default 2^18
    const int M = 2*N - 1;                                  // corr length
    const int Mp2 = next_pow2(M);                           // FFT size (power-2)

    std::cout<<"Signal length N           : "<<N<<"\n"
             <<"Correlation sequence size : "<<M<<"\n"
             <<"FFT size (next pow-2)      : "<<Mp2<<"\n\n";

    // Host input signals (white noise)
    std::vector<float> h_x(N), h_y(N);
    std::mt19937 rng(42); std::uniform_real_distribution<float> dist(-1.f,1.f);
    for(float& v:h_x) v=dist(rng);
    for(float& v:h_y) v=dist(rng);

    // Device buffers
    float *d_x, *d_y, *d_corr_time, *d_corr_fft;
    CUDA_CHECK(hipMalloc(&d_x,           sizeof(float)*N));
    CUDA_CHECK(hipMalloc(&d_y,           sizeof(float)*N));
    CUDA_CHECK(hipMalloc(&d_corr_time,   sizeof(float)*M));
    CUDA_CHECK(hipMalloc(&d_corr_fft ,   sizeof(float)*M));

    // Common H2D copy
    hipEvent_t eH0,eH1; hipEventCreate(&eH0); hipEventCreate(&eH1);
    hipEventRecord(eH0);
    CUDA_CHECK(hipMemcpy(d_x,h_x.data(),sizeof(float)*N,hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_y,h_y.data(),sizeof(float)*N,hipMemcpyHostToDevice));
    hipEventRecord(eH1); CUDA_CHECK(hipDeviceSynchronize());
    float h2d_ms; hipEventElapsedTime(&h2d_ms,eH0,eH1);
    hipEventDestroy(eH0); hipEventDestroy(eH1);

    // 1) Time-domain O(N²)
    Times tTime;
    {
        hipEvent_t k0,k1,c0,c1; hipEventCreate(&k0); hipEventCreate(&k1);
        hipEventCreate(&c0); hipEventCreate(&c1);

        int blocks = (M + THREADS - 1)/THREADS;
        hipEventRecord(k0);
        xcorr_naive<<<blocks,THREADS>>>(d_x,d_y,d_corr_time,N);
        hipEventRecord(k1);

        std::vector<float> h_corr_time(M);
        hipEventRecord(c0);
        CUDA_CHECK(hipMemcpy(h_corr_time.data(),d_corr_time,
                              sizeof(float)*M,hipMemcpyDeviceToHost));
        hipEventRecord(c1); CUDA_CHECK(hipDeviceSynchronize());

        hipEventElapsedTime(&tTime.ker ,k0,k1);
        hipEventElapsedTime(&tTime.d2h,c0,c1);
        tTime.h2d = h2d_ms;
        
        hipEventDestroy(k0); hipEventDestroy(k1);
        hipEventDestroy(c0); hipEventDestroy(c1);
    }

    // 2) FFT  O(N log N)
    Times tFFT;
    {
        // Complex scratch arrays (zero-padded)
        hipfftComplex *d_X,*d_Y,*d_Z;
        CUDA_CHECK(hipMalloc(&d_X,sizeof(hipfftComplex)*Mp2));
        CUDA_CHECK(hipMalloc(&d_Y,sizeof(hipfftComplex)*Mp2));
        CUDA_CHECK(hipMalloc(&d_Z,sizeof(hipfftComplex)*Mp2));
        CUDA_CHECK(hipMemset(d_X,0,sizeof(hipfftComplex)*Mp2));
        CUDA_CHECK(hipMemset(d_Y,0,sizeof(hipfftComplex)*Mp2));

        // Copy real -> complex (imag=0)
        CUDA_CHECK(hipMemcpy2D(d_X,sizeof(hipfftComplex),
                                d_x,sizeof(float),
                                sizeof(float),N,
                                hipMemcpyDeviceToDevice));
        CUDA_CHECK(hipMemcpy2D(d_Y,sizeof(hipfftComplex),
                                d_y,sizeof(float),
                                sizeof(float),N,
                                hipMemcpyDeviceToDevice));

        hipfftHandle plan; CUFFT_CHECK(hipfftPlan1d(&plan,Mp2,HIPFFT_C2C,1));

        hipEvent_t k0,k1,c0,c1; hipEventCreate(&k0); hipEventCreate(&k1);
        hipEventCreate(&c0); hipEventCreate(&c1);

        hipEventRecord(k0);
        // Forward FFTs
        CUFFT_CHECK(hipfftExecC2C(plan,d_X,d_X,HIPFFT_FORWARD));
        CUFFT_CHECK(hipfftExecC2C(plan,d_Y,d_Y,HIPFFT_FORWARD));
        // Multiply X · conj(Y)
        int blocksMul = (Mp2 + THREADS - 1)/THREADS;
        cmul_conj<<<blocksMul,THREADS>>>(d_X,d_Y,d_Z,Mp2);
        // Inverse FFT
        CUFFT_CHECK(hipfftExecC2C(plan,d_Z,d_Z,HIPFFT_BACKWARD));

        // Scale & copy first M real samples
        float scale = 1.0f / Mp2;
        int blocksSC = (M + THREADS - 1)/THREADS;
        scaleCopyKernel<<<blocksSC,THREADS>>>(d_corr_fft,d_Z,scale,M);
        hipEventRecord(k1);

        // Copy back
        std::vector<float> h_corr_fft(M);
        hipEventRecord(c0);
        CUDA_CHECK(hipMemcpy(h_corr_fft.data(),d_corr_fft,
                              sizeof(float)*M,hipMemcpyDeviceToHost));
        hipEventRecord(c1); CUDA_CHECK(hipDeviceSynchronize());

        hipEventElapsedTime(&tFFT.ker ,k0,k1);
        hipEventElapsedTime(&tFFT.d2h,c0,c1);
        tFFT.h2d = h2d_ms;

        hipEventDestroy(k0); hipEventDestroy(k1);
        hipEventDestroy(c0); hipEventDestroy(c1);
        hipfftDestroy(plan);
        hipFree(d_X); hipFree(d_Y); hipFree(d_Z);
    }

    // Timings
    auto line=[&](const char* n,const Times& t){
        std::cout<<std::setw(13)<<std::left<<n
                 <<" H2D "<<std::setw(7)<<t.h2d
                 <<" Kern "<<std::setw(9)<<t.ker
                 <<" D2H "<<std::setw(7)<<t.d2h
                 <<" Total "<<t.total()<<" ms\n";
    };
    std::cout<<std::fixed<<std::setprecision(3);
    std::cout<<"\n               H2D      Kern        D2H     Total\n";
    line("Time-domain", tTime);
    line("FFT        ", tFFT);

    // Cleanup
    hipFree(d_x); hipFree(d_y);
    hipFree(d_corr_time); hipFree(d_corr_fft);
    return 0;
}